#include <hip/hip_runtime.h>
//#include <cutil.h>
#include <iostream>
#include <ostream>
#include <fstream>
//#include "/home/yusuke/NVIDIA_GPU_Computing_SDK/C/common/inc/cutil.h"
using namespace std;
	
#define CASENAME "Test31"

#define BLOCKSIZEX 128
#define BLOCKSIZEY 1
#define BLOCKSIZEZ 1
#define BLOCKSIZELRX 64
#define BLOCKSIZELRY 1
#define BLOCKSIZELRZ 1
#define XDIM 128
#define YDIM 128
#define ZDIM 64
#define TMAX 100
#define STARTF 0

#define OBSTR1 4.f
#define OBSTX1 31.5f
#define OBSTY1 31.5f
#define OBSTZ1 15.5f

#define OBSTR2 4.f
#define OBSTX2 63.5f
#define OBSTY2 31.5f
#define OBSTZ2 31.5f

#define RE 100.f//2000.f//100.f;
#define UMAX 0.08f
#define METHOD "SINGLE" //SINGLE,HYB,TEXT,SHARED,CACHE
#define SmagLES "NO" //YES,NO
#define MODEL "BGK" //BGK,MRT,STREAM
#define ZPERIODIC "NO"
#define CS 0.04f
//#define CHARLENGTH = XDIM-2.f;
//#define BLOCKSIZE 16;
//int const XDIM = 32;
//int const YDIM = 32;

#include <sys/time.h>
#include <time.h>

/*
Image List:
0  fluid
1  BB
2
3  DirichletWest(simple)
10 BB(force)
13 DirichletWest_Reg
14 NeumannEast_Reg
15 DirichletNorth_Reg
16 DirichletSouth_Reg
21 ysymmetry_top
22 ysymmetry_bot
23 zsymmetry_top
24 zsymmetry_bot
25 xsymmetry_top
26 xsymmetry_bot
*/
inline __device__ int ImageFcn(float x, float y, float z){
//	if(((x-OBSTX1)*(x-OBSTX1)+(y-OBSTY1)*(y-OBSTY1))<OBSTR1*OBSTR1)
//		return 10;
//	else if(((x-OBSTX2)*(x-OBSTX2)+(y-OBSTY2)*(y-OBSTY2))<OBSTR2*OBSTR2)
//		return 10;
	//if(((x-OBSTX)*(x-OBSTX)+(y-OBSTY1)*(y-OBSTY1))<OBSTR1*OBSTR1)
//	if(((x-OBSTX1)*(x-OBSTX1)+(y-OBSTY1)*(y-OBSTY1)+(z-OBSTZ1)*(z-OBSTZ1))<OBSTR1*OBSTR1)
//	{
//		return 10;
//	}
//	else
//	//if(y < 0.1f || z < 0.1f || (XDIM-x) < 0.1f || (YDIM-y) < 0.1f || (ZDIM-z) < 0.1f)
//	if(y < 17.5f || z < 17.5f || y > 46.5f || z > 46.5f)
//		return 1;
//	else if(x < 17.5f)
//		return 13;
//	else if(x > 78.5f)
//		return 14;
//	else
    
    if(abs(x-OBSTX1) < OBSTR1 && abs(y-OBSTY1) < OBSTR1)
        return 10;
    else
		return 0;
}

inline __device__ int ImageFcn(int x, int y, int z){
    int value = 0;
//Cylinder
//	if(((x-OBSTX1)*(x-OBSTX1)+(y-OBSTY1)*(y-OBSTY1))<OBSTR1*OBSTR1)
//		value = 10;
//	else if(((x-OBSTX2)*(x-OBSTX2)+(y-OBSTY2)*(y-OBSTY2))<OBSTR2*OBSTR2)
//		value = 10;
//Sphere
//	if(((x-OBSTX1)*(x-OBSTX1)+(y-OBSTY1)*(y-OBSTY1)+(z-OBSTZ1)*(z-OBSTZ1))<OBSTR1*OBSTR1)
//	{
////		if(z == 0 || z == ZDIM-1)
////		return 1;
////		else
//		return 10;
//	}
//	if(z == 0)
//		value = 0;
//	else if(z == ZDIM-1)
//		value = 0;

//    if(abs(x-OBSTX1) < OBSTR1 && abs(y-OBSTY1) < OBSTR1)
//        value = 10;
//	else if(y == 0)
//		value = 200;//22;
//	else if(y == YDIM-1)
//		value = 100;
//	else if(x == 0)
//		value = 26;
//	else if(x == XDIM-1)
//		value = 25;
//	else if(z == 0)
//		value = 0;
//	else if(z == ZDIM-1)
//		value = 0;

    //return value;

//Lid Driven Cavity
//	if(y == 0 || y == YDIM-1 || z == 0 || z == ZDIM-1)
//		value = 1;
//	else if(x == XDIM-2 || y == 1 || y == YDIM-2 || z == 1 || z == ZDIM-2)
//		return 1;
//	else if(x == 0)
//		return 1;
    
//	if(abs(x-OBSTX1) < OBSTR1 && abs(y-OBSTY1) < OBSTR1)
//        value = 10;
	if(y == 0)
		value = 200;//22;
	else if(y == YDIM-1)
		value = 100;
	else if(x == 0)
		value = 1;
	else if(x == XDIM-1)
		value = 1;
//	else if(x == 0)
//		return 53;
//	else if(x == XDIM-1)
//		return 54;
	return value;
}

inline __device__ float PoisProf (float x){
	float radius = (YDIM-1-1)*0.5f;
	float result = -1.0f*(((1.0f-(x-0.5f)/radius))*((1.0f-(x-0.5f)/radius))-1.0f);
	return (result);
//	return 1.f;
}

__device__ void DirichletWest(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
//	if(y == 0){
//		f2 = f4;
//		f6 = f7;
//		f11 = f13;
//		f16 = f18;
//	}
//	else if(y == YDIM-1){
//		f4 = f2;
//		f7 = f6;
//		f13 = f11;
//		f18 = f16;
//	}
//	if(z == 0){
//		f9  = f14;
//		f10 = f15;
//		f11 = f16;
//		f12 = f17;
//		f13 = f18;			
//	}    
//	else if(z == ZDIM-1){
//		f14 = f9;
//		f15 = f10;
//		f16 = f11;
//		f17 = f12;
//		f18 = f13;
//	}
	if(y == 0 && z == 0){
		f2 = f4;
        f13=f18;
        f11=f18;
        f16=f18;
		f6 =f7;
        f9 =f14;
        f12=f17;
	}
	else if(y == 0 && z == ZDIM-1){
		f4 = f2;
        f11=f13;
        f18=f13;
        f16=f13;
		f6 =f7;
        f14=f9;
        f17=f12;
	}
	else if(y == YDIM-1 && z == 0){
		f4 = f2;
        f11=f16;
        f18=f16;
        f13=f16;
		f7 =f6;
        f9 =f14;
        f12=f17;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4 = f2;
        f16=f11;
        f18=f11;
        f13=f11;
		f7 =f6;
        f14=f9;
        f17=f12;
	}
    else{
	if(y == 0){
        f2 = f4;
        f11=f13;
        f16=f18;
        f8 = f5;
    }
	else if(y == YDIM-1){
         f4=f2 ;
        f13=f11;
        f18=f16;
         f5=f8 ;
    }
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
		f18 = f13;
	}
    }
	float u,v,w;//,rho;
    u = UMAX;//*PoisProf(zcoord)*1.5;
    v = 0.0f;
	w = 0.0f;
    
//	float rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i
//    float usqr = u*u+v*v+w*w;

	f1 = fma(0.0555555556f,6.0f*u,f3);//0.0555555556f*(6.0f*u)+f3;//-0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);;
	f5 = fma(0.0277777778f,6.0f*(u+v),f7 );// -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f8 = fma(0.0277777778f,6.0f*(u-v),f6 );// -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f10= fma(0.0277777778f,6.0f*(u+w),f17);//-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f15= fma(0.0277777778f,6.0f*(u-w),f12);//-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
////	f0 = 1.0f/3.0f*(rho-1.5f*usqr);
//	f1 = 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
////	f2 = 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
////	f3 = 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
////	f4 = 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
//	f5 = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
////	f6 = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
////	f7 = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
//	f8 = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
////	f9 = 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
//	f10= 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
////	f11= 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr);
////	f12= 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
////	f13= 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr);
////	f14= 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
//	f15= 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
////	f16= 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
////	f17= 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
////	f18= 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);
}

__device__ void DirichletWest_Reg(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0){
		f2 = f4;
		f6 = f7;
		f11 = f13;
		f16 = f18;
	}
	else if(y == YDIM-1){
		f4 = f2;
		f7 = f6;
		f13 = f11;
		f18 = f16;
	}
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
		f12 = f17;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
		f17 = f12;
		f18 = f13;
	}
	float u,v,w;//,rho;
    u = UMAX;//*PoisProf(y)*1.5;
    v = 0.0f;//0.0;
	w = 0.0f;
//	float rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i
//	float u2 = u*u;
//	float v2 = v*v;
//	float w2 = w*w;
//	float usqr = u2+v2+w2;

//	f1 =(0.166666667f*u)+
//		(f3-(-(0.166666667f*u)));
	f1 = f3+0.33333333f*u;
//	f5 =(0.0833333333f*( u+v))+
//		(f7-(0.0833333333f*(-u-v)));
	f5 = f7+0.166666667f*(u+v);
//	f8 =(0.0833333333f*( u-v  ))+
//		(f6-(0.0833333333f*(-u+v  )));
	f8 = f6+0.166666667f*(u-v);
//	f10=(0.0833333333f*( u+w))+
//		(f17-(0.0833333333f*(-u-w)));
	f10= f17+0.166666667f*(u+w);
//	f15=(0.0833333333f*( u-w))+
//		(f12-(0.0833333333f*(-u+w)));
	f15= f12+0.166666667f*(u-w);
		
//	f1 =(0.1031746045f*rho+  -0.0231796391f*usqr+ (0.166666667f*u)   + 0.16666667f*u2)+
//		(f3-(0.1031746045f*rho+  -0.0231796391f*usqr+-(0.166666667f*u)   + 0.16666667f*u2));
//	f5 =(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u+v  +u2+(v2-w2))+  0.25f*u*v)+
//		(f7-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u-v  +u2+(v2-w2))+  0.25f*u*v));
//	f8 =(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u-v  +u2+(v2-w2))+ -0.25f*u*v)+
//		(f6-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u+v  +u2+(v2-w2))+ -0.25f*u*v));
//	f10=(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u+w  +u2+(v2-w2))+  0.25f*u*w)+
//		(f17-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u-w  +u2+(v2-w2))+  0.25f*u*w));
//	f15=(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u-w  +u2+(v2-w2))+ -0.25f*u*w)+
//		(f12-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u+w  +u2+(v2-w2))+ -0.25f*u*w));

//	float PI11 = f1 +f3 +f5 +f6 +f7 +f8 +f10+f12+f15+f17;
//	float PI22 = f2 +f4 +f5 +f6 +f7 +f8 +f11+f13+f16+f18;
//	float PI33 = f9 +f10+f11+f12+f13+f14+f15+f16+f17+f18;

}


void __device__ DirichletWest_Regularized(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0 && z == 0){
		f2 = f4;
        f13=f18;
        f11=f18;
        f16=f18;
		f6 =f7;
        f9 =f14;
        f12=f17;
	}
	else if(y == 0 && z == ZDIM-1){
		f4 = f2;
        f11=f13;
        f18=f13;
        f16=f13;
		f6 =f7;
        f14=f9;
        f17=f12;
	}
	else if(y == YDIM-1 && z == 0){
		f4 = f2;
        f11=f16;
        f18=f16;
        f13=f16;
		f7 =f6;
        f9 =f14;
        f12=f17;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4 = f2;
        f16=f11;
        f18=f11;
        f13=f11;
		f7 =f6;
        f14=f9;
        f17=f12;
	}
    else{
	if(y == 0){
        f2 = f4;
        f11=f13;
        f16=f18;
        f8 = f5;
    }
	else if(y == YDIM-1){
         f4=f2 ;
        f13=f11;
        f18=f16;
         f5=f8 ;
    }
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
		f18 = f13;
	}
    }

	float PI11 = 0;
	float PI12 = 0;
	float PI22 = 0;
	float PI33 = 0;
	float PI13 = 0;
	float PI23 = 0;
	float u;//,v;//,w;//,rho;
    u = UMAX;//*PoisProf(z)*1.5;
    //v = 0.0f;
	//w = 0.0f;
    float usqr = u*u;//+v*v+w*w;
	float rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i
    
    float feq0  = 0.3333333333f*(rho-1.5f*usqr);
    float feq1  = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
    float feq2  = 0.0555555556f*(rho                -1.5f*usqr);
    float feq3  = 0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
    float feq4  = 0.0555555556f*(rho                -1.5f*usqr);
    float feq9  = 0.0555555556f*(rho                -1.5f*usqr);
    float feq14 = 0.0555555556f*(rho                -1.5f*usqr);
    float feq5  = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq6  = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq7  = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq8  = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq10 = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq11 = 0.0277777778f*(rho                         -1.5f*usqr);
    float feq12 = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq13 = 0.0277777778f*(rho                         -1.5f*usqr);
    float feq15 = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq16 = 0.0277777778f*(rho                         -1.5f*usqr);
    float feq17 = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq18 = 0.0277777778f*(rho                         -1.5f*usqr);

//    float feq0  = 0.3333333333f*(rho-1.5f*usqr);
//    float feq1  = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
//    float feq2  = 0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
//    float feq3  = 0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
//    float feq4  = 0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
//    float feq5  = 0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
//    float feq6  = 0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
//    float feq7  = 0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
//    float feq8  = 0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
//    float feq9  = 0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
//    float feq10 = 0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
//    float feq11 = 0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(v+w)-1.5f*usqr);
//    float feq12 = 0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
//    float feq13 = 0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(-v+w)-1.5f*usqr);
//    float feq14 = 0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
//    float feq15 = 0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
//    float feq16 = 0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
//    float feq17 = 0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
//    float feq18 = 0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);

	f1 = feq1 +f3 -feq3 ;
	f5 = feq5 +f7 -feq7 ;
	f8 = feq8 +f6 -feq6 ;
	f10= feq10+f17-feq17;
	f15= feq15+f12-feq12;

    PI11 = (f1-feq1)+(f3-feq3)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17);
    PI22 = (f2-feq2)+(f4-feq4)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
    PI33 = (f9-feq9)+(f14-feq14)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
    PI12 = (f5-feq5)+(f7-feq7)-(f6-feq6)-(f8-feq8);
    PI13 = (f10-feq10)+(f17-feq17)-(f12-feq12)-(f15-feq15);
    PI23 = (f11-feq11)+(f18-feq18)-(f13-feq13)-(f16-feq16);

    f0  = feq0 +1.5f  *((     -0.333333333f)*PI11                         +(     -0.333333333f)*PI22+(     -0.333333333f)*PI33)  ;
    f1  = feq1 +0.25f *((      0.666666667f)*PI11                         +(     -0.333333333f)*PI22+(     -0.333333333f)*PI33)  ;
    f2  = feq2 +0.25f *((     -0.333333333f)*PI11                         +(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f3  = feq3 +0.25f *((      0.666666667f)*PI11                         +(     -0.333333333f)*PI22+(     -0.333333333f)*PI33)  ;
    f4  = feq4 +0.25f *((     -0.333333333f)*PI11                         +(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f5  = feq5 +0.125f*((      0.666666667f)*PI11+2.0f*( PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f6  = feq6 +0.125f*((      0.666666667f)*PI11+2.0f*(-PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f7  = feq7 +0.125f*((      0.666666667f)*PI11+2.0f*( PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f8  = feq8 +0.125f*((      0.666666667f)*PI11+2.0f*(-PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f9  = feq9 +0.25f *((     -0.333333333f)*PI11                         +(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f10 = feq10+0.125f*((      0.666666667f)*PI11+2.0f*(     + PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f11 = feq11+0.125f*((     -0.333333333f)*PI11+2.0f*(           + PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
    f12 = feq12+0.125f*((      0.666666667f)*PI11+2.0f*(     +-PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f13 = feq13+0.125f*((     -0.333333333f)*PI11+2.0f*(           +-PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
    f14 = feq14+0.25f *((     -0.333333333f)*PI11                         +(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f15 = feq15+0.125f*((      0.666666667f)*PI11+2.0f*(     +-PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f16 = feq16+0.125f*((     -0.333333333f)*PI11+2.0f*(           +-PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
    f17 = feq17+0.125f*((      0.666666667f)*PI11+2.0f*(     + PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f18 = feq18+0.125f*((     -0.333333333f)*PI11+2.0f*(           + PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
		
}



void __device__ NeumannEast_Regularized(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0 && z == 0){
		f2  = f4;
		f13 = f18;			
		f11 = f18;
		f16 = f18;
		f5  = f8;
		f9  = f14;
		f10 = f15;
	}
	else if(y == 0 && z == ZDIM-1){
		f2  = f4;
		f11 = f13;
		f18 = f13;
		f16 = f13;
		f5  = f8;
		f14 = f9;
		f15 = f10;
	}
	else if(y == YDIM-1 && z == 0){
		f4  = f2;
		f18 = f16;
		f11 = f16;
		f13 = f16;	
		f8  = f5;
		f9  = f14;
		f10 = f15;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4  = f2;
		f13 = f11;
		f16 = f11;
		f18 = f11;
		f8  = f5;
		f14 = f9;
		f15 = f10;
	}

	else{
	if(y == 0){
		f2 = f4;
		f11 = f13;
		f16 = f18;
		f5 = f8;
	}
	else if(y == YDIM-1){
		f4 = f2;
		f13 = f11;
		f18 = f16;
		f8 = f5;
	}
	else if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
		f18 = f13;
	}
	}

	float PI11 = 0;
	float PI12 = 0;
	float PI22 = 0;
	float PI33 = 0;
	float PI13 = 0;
	float PI23 = 0;

	float u;//,v;//,w;//,rho;
	float rho = 1.0f;
    //v = 0.0f;
	//w = 0.0f;
	u = -rho+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f1+f8+f5+f10+f15)); //D2Q9i
    float usqr = u*u;//+v*v+w*w;
    
    float feq0  = 0.3333333333f*(rho-1.5f*usqr);
    float feq1  = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
    float feq2  = 0.0555555556f*(rho                -1.5f*usqr);
    float feq3  = 0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
    float feq4  = 0.0555555556f*(rho                -1.5f*usqr);
    float feq9  = 0.0555555556f*(rho                -1.5f*usqr);
    float feq14 = 0.0555555556f*(rho                -1.5f*usqr);
    float feq5  = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq6  = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq7  = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq8  = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq10 = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq11 = 0.0277777778f*(rho                         -1.5f*usqr);
    float feq12 = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq13 = 0.0277777778f*(rho                         -1.5f*usqr);
    float feq15 = 0.0277777778f*(rho+3.0f*( u)+4.5f*( u)*( u)-1.5f*usqr);
    float feq16 = 0.0277777778f*(rho                         -1.5f*usqr);
    float feq17 = 0.0277777778f*(rho+3.0f*(-u)+4.5f*(-u)*(-u)-1.5f*usqr);
    float feq18 = 0.0277777778f*(rho                         -1.5f*usqr);
    
//    float feq0  = 0.3333333333f*(rho-1.5f*usqr);
//    float feq1  = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
//    float feq2  = 0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
//    float feq3  = 0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
//    float feq4  = 0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
//    float feq9  = 0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
//    float feq14 = 0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
//    float feq5  = 0.0277777778f*(rho+3.0f*( u+v)+4.5f*( u+v)*( u+v)-1.5f*usqr);
//    float feq6  = 0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
//    float feq7  = 0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
//    float feq8  = 0.0277777778f*(rho+3.0f*( u-v)+4.5f*( u-v)*( u-v)-1.5f*usqr);
//    float feq10 = 0.0277777778f*(rho+3.0f*( u+w)+4.5f*( u+w)*( u+w)-1.5f*usqr);
//    float feq11 = 0.0277777778f*(rho+3.0f*( v+w)+4.5f*( v+w)*( v+w)-1.5f*usqr);
//    float feq12 = 0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
//    float feq13 = 0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(-v+w)-1.5f*usqr);
//    float feq15 = 0.0277777778f*(rho+3.0f*( u-w)+4.5f*( u-w)*( u-w)-1.5f*usqr);
//    float feq16 = 0.0277777778f*(rho+3.0f*( v-w)+4.5f*( v-w)*( v-w)-1.5f*usqr);
//    float feq17 = 0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
//    float feq18 = 0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);

	f3 = feq3 +f1 -feq1 ;
	f7 = feq7 +f5 -feq5 ;
	f6 = feq6 +f8 -feq8 ;
	f17= feq17+f10-feq10;
	f12= feq12+f15-feq15;

    PI11 = (f1-feq1)+(f3-feq3)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17);
    PI22 = (f2-feq2)+(f4-feq4)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
    PI33 = (f9-feq9)+(f14-feq14)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
    PI12 = (f5-feq5)+(f7-feq7)-(f6-feq6)-(f8-feq8);
    PI13 = (f10-feq10)+(f17-feq17)-(f12-feq12)-(f15-feq15);
    PI23 = (f11-feq11)+(f18-feq18)-(f13-feq13)-(f16-feq16);

    f0  = feq0 +1.5f  *((     -0.333333333f)*PI11                         +(     -0.333333333f)*PI22+(     -0.333333333f)*PI33)  ;
    f1  = feq1 +0.25f *((      0.666666667f)*PI11                         +(     -0.333333333f)*PI22+(     -0.333333333f)*PI33)  ;
    f2  = feq2 +0.25f *((     -0.333333333f)*PI11                         +(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f3  = feq3 +0.25f *((      0.666666667f)*PI11                         +(     -0.333333333f)*PI22+(     -0.333333333f)*PI33)  ;
    f4  = feq4 +0.25f *((     -0.333333333f)*PI11                         +(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f5  = feq5 +0.125f*((      0.666666667f)*PI11+2.0f*( PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f6  = feq6 +0.125f*((      0.666666667f)*PI11+2.0f*(-PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f7  = feq7 +0.125f*((      0.666666667f)*PI11+2.0f*( PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f8  = feq8 +0.125f*((      0.666666667f)*PI11+2.0f*(-PI12            )+(      0.666666667f)*PI22+(     -0.333333333f)*PI33)  ;
    f9  = feq9 +0.25f *((     -0.333333333f)*PI11                         +(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f10 = feq10+0.125f*((      0.666666667f)*PI11+2.0f*(     + PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f11 = feq11+0.125f*((     -0.333333333f)*PI11+2.0f*(           + PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
    f12 = feq12+0.125f*((      0.666666667f)*PI11+2.0f*(     +-PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f13 = feq13+0.125f*((     -0.333333333f)*PI11+2.0f*(           +-PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
    f14 = feq14+0.25f *((     -0.333333333f)*PI11                         +(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f15 = feq15+0.125f*((      0.666666667f)*PI11+2.0f*(     +-PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f16 = feq16+0.125f*((     -0.333333333f)*PI11+2.0f*(           +-PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
    f17 = feq17+0.125f*((      0.666666667f)*PI11+2.0f*(     + PI13      )+(     -0.333333333f)*PI22+(      0.666666667f)*PI33)  ;
    f18 = feq18+0.125f*((     -0.333333333f)*PI11+2.0f*(           + PI23)+(      0.666666667f)*PI22+(      0.666666667f)*PI33)  ;
			
}


__device__ void NeumannEast(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0 && z == 0){
		f2  = f4;
		f13 = f18;			
		f11 = f18;
		f16 = f18;
		f5  = f8;
		f9  = f14;
		f10 = f15;
	}
	else if(y == 0 && z == ZDIM-1){
		f2  = f4;
		f11 = f13;
		f18 = f13;
		f16 = f13;
		f5  = f8;
		f14 = f9;
		f15 = f10;
	}
	else if(y == YDIM-1 && z == 0){
		f4  = f2;
		f18 = f16;
		f11 = f16;
		f13 = f16;	
		f8  = f5;
		f9  = f14;
		f10 = f15;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4  = f2;
		f13 = f11;
		f16 = f11;
		f18 = f11;
		f8  = f5;
		f14 = f9;
		f15 = f10;
	}

	else{
	if(y == 0){
		f2 = f4;
//		f6 = f7;
		f11 = f13;
		f16 = f18;

		f5 = f8;
	}
	else if(y == YDIM-1){
		f4 = f2;
//		f7 = f6;
		f13 = f11;
		f18 = f16;

		f8 = f5;
	}
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
//		f12 = f17;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
//		f17 = f12;
		f18 = f13;
	}
	}

	float u,v,w;//,rho;
	float rho = 1.0f;
    v = 0.0f;
	w = 0.0f;
	u = -rho+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f1+f8+f5+f10+f15)); //D2Q9i
	float u2 = u*u;
	float v2 = v*v;
	float w2 = w*w;
	float usqr = u2+v2+w2;

//	f3 = f1 -0.333333333f*u;
//	f7 = f5 -0.166666667f*(u+v);
//	f6 = f8 -0.166666667f*(u-v);
//	f17= f10-0.166666667f*(u+w);
//	f12= f15-0.166666667f*(u-w);
	f0 = 1.0f/3.0f*(rho-1.5f*usqr);
	f1 = 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f2 = 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f3 = 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f4 = 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f5 = 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f6 = 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f7 = 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f8 = 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f9 = 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
	f10= 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
	f11= 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr);
	f12= 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
	f13= 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr);
	f14= 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
	f15= 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
	f16= 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
	f17= 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f18= 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);


}


__device__ void NeumannEast_Reg(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0 && z == 0){
		f2  = f4;
		f13 = f18;			
		f11 = f18;
		f16 = f18;
		f5  = f8;
		f9  = f14;
		f10 = f15;
	}
	else if(y == 0 && z == ZDIM-1){
		f2  = f4;
		f11 = f13;
		f18 = f13;
		f16 = f13;
		f5  = f8;
		f14 = f9;
		f15 = f10;
	}
	else if(y == YDIM-1 && z == 0){
		f4  = f2;
		f18 = f16;
		f11 = f16;
		f13 = f16;	
		f8  = f5;
		f9  = f14;
		f10 = f15;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4  = f2;
		f13 = f11;
		f16 = f11;
		f18 = f11;
		f8  = f5;
		f14 = f9;
		f15 = f10;
	}

	else{
	if(y == 0){
		f2 = f4;
//		f6 = f7;
		f11 = f13;
		f16 = f18;

		f5 = f8;
	}
	else if(y == YDIM-1){
		f4 = f2;
//		f7 = f6;
		f13 = f11;
		f18 = f16;

		f8 = f5;
	}
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
//		f12 = f17;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
//		f17 = f12;
		f18 = f13;
	}
	}

	float u,v,w;//,rho;
	float rho = 1.0f;
    v = 0.0f;
	w = 0.0f;
	u = -rho+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f1+f8+f5+f10+f15)); //D2Q9i
//	float u2 = u*u;
//	float v2 = v*v;
//	float w2 = w*w;
//	float usqr = u2+v2+w2;

	f3 = f1 -0.333333333f*u;
	f7 = f5 -0.166666667f*(u+v);
	f6 = f8 -0.166666667f*(u-v);
	f17= f10-0.166666667f*(u+w);
	f12= f15-0.166666667f*(u-w);


//	f3 =(0.1031746045f*rho+  -0.0231796391f*usqr+-(0.166666667f*u)   + 0.16666667f*u2)+
//		(f1-(0.1031746045f*rho+  -0.0231796391f*usqr+ (0.166666667f*u)   + 0.16666667f*u2));
//	f7 =(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u-v  +u2+(v2-w2))+  0.25f*u*v)+
//		(f5-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u+v  +u2+(v2-w2))+  0.25f*u*v));
//	f6 =(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u+v  +u2+(v2-w2))+ -0.25f*u*v)+
//		(f8-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u-v  +u2+(v2-w2))+ -0.25f*u*v));
//	f17=(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u-w  +u2+(v2-w2))+  0.25f*u*w)+
//		(f10-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u+w  +u2+(v2-w2))+  0.25f*u*w));
//	f12=(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u+w  +u2+(v2-w2))+ -0.25f*u*w)+
//		(f15-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u-w  +u2+(v2-w2))+ -0.25f*u*w));


//	f1 =(0.1031746045f*rho+  -0.0231796391f*usqr+ (0.166666667f*u)   + 0.16666667f*u2)+
//		(f3-(0.1031746045f*rho+  -0.0231796391f*usqr+-(0.166666667f*u)   + 0.16666667f*u2));
//	f5 =(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u+v  +u2+(v2-w2))+  0.25f*u*v)+
//		(f7-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u-v  +u2+(v2-w2))+  0.25f*u*v));
//	f8 =(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u-v  +u2+(v2-w2))+ -0.25f*u*v)+
//		(f6-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u+v  +u2+(v2-w2))+ -0.25f*u*v));
//	f10=(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u+w  +u2+(v2-w2))+  0.25f*u*w)+
//		(f17-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u-w  +u2+(v2-w2))+  0.25f*u*w));
//	f15=(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*( u-w  +u2+(v2-w2))+ -0.25f*u*w)+
//		(f12-(0.0158730149f*rho+  0.00579491071f*usqr+ 0.0833333333f*(-u+w  +u2+(v2-w2))+ -0.25f*u*w));

//	float PI11 = f1 +f3 +f5 +f6 +f7 +f8 +f10+f12+f15+f17;
//	float PI22 = f2 +f4 +f5 +f6 +f7 +f8 +f11+f13+f16+f18;
//	float PI33 = f9 +f10+f11+f12+f13+f14+f15+f16+f17+f18;

}

__device__ void DirichletNorth_Reg(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
//	if(x == 0){
//		f2 = f4;
//		f6 = f7;
//		f11 = f13;
//		f16 = f18;
//	}
//	else if(x == XDIM-1){
//		f4 = f2;
//		f7 = f6;
//		f13 = f11;
//		f18 = f16;
//	}
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
		f12 = f17;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
		f17 = f12;
		f18 = f13;
	}
	float u,v,w;//,rho;
    u = UMAX;
    v = 0.0f;//0.0;
	w = 0.0f;
//	float rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i
//	float u2 = u*u;
//	float v2 = v*v;
//	float w2 = w*w;
//	float usqr = u2+v2+w2;

//	f1 =(0.166666667f*u)+
//		(f3-(-(0.166666667f*u)));
	f4 = f2-0.33333333f*v;
//	f5 =(0.0833333333f*( u+v))+
//		(f7-(0.0833333333f*(-u-v)));
	f7 = f5-0.166666667f*(u+v);
//	f8 =(0.0833333333f*( u-v  ))+
//		(f6-(0.0833333333f*(-u+v  )));
	f8 = f6+0.166666667f*(u-v);
//	f10=(0.0833333333f*( u+w))+
//		(f17-(0.0833333333f*(-u-w)));
	f13= f16-0.166666667f*(v-w);
//	f15=(0.0833333333f*( u-w))+
//		(f12-(0.0833333333f*(-u+w)));
	f18= f11-0.166666667f*(v+w);
	
//
//float feq0 = 0.1904761791f*rho+-0.597127747f*usqr                     
//float feq1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u    + 0.055555556f*(2.f*u*u-(v*v+w*w));
//float feq2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v    +-0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
//float feq3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u    + 0.055555556f*(2.f*u*u-(v*v+w*w));
//float feq4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v    +-0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
//float feq5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
//float feq6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
//float feq7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
//float feq8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
//float feq9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;   +-0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                           
//float feq10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
//float feq11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w)+-0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
//float feq12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
//float feq13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w)+-0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
//float feq14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w    +-0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                           
//float feq15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w) + 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
//float feq16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w) +-0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
//float feq17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w) + 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
//float feq18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w) +-0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;
//



//	float PI11 = f1 +f3 +f5 +f6 +f7 +f8 +f10+f12+f15+f17;
//	float PI22 = f2 +f4 +f5 +f6 +f7 +f8 +f11+f13+f16+f18;
//	float PI33 = f9 +f10+f11+f12+f13+f14+f15+f16+f17+f18;

}
__device__ void DirichletSouth_Reg(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
//	if(x == 0){
//		f2 = f4;
//		f6 = f7;
//		f11 = f13;
//		f16 = f18;
//	}
//	else if(x == XDIM-1){
//		f4 = f2;
//		f7 = f6;
//		f13 = f11;
//		f18 = f16;
//	}
	if(z == 0){
		f9  = f14;
		f10 = f15;
		f11 = f16;
		f12 = f17;
		f13 = f18;			
	}    
	else if(z == ZDIM-1){
		f14 = f9;
		f15 = f10;
		f16 = f11;
		f17 = f12;
		f18 = f13;
	}
	float u,v,w;//,rho;
    u = UMAX;
    v = 0.0f;//0.0;
	w = 0.0f;
//	float rho = u+(f0+f2+f4+f9+f11+f13+f14+f16+f18+2.0f*(f3+f6+f7+f12+f17)); //D2Q9i
//	float u2 = u*u;
//	float v2 = v*v;
//	float w2 = w*w;
//	float usqr = u2+v2+w2;

	f2 = f4 +0.33333333f*v;
	f5 = f7 +0.166666667f*(u+v);
	f6 = f8 -0.166666667f*(u-v);
	f16= f13+0.166666667f*(v-w);
	f11= f18+0.166666667f*(v+w);
	
//
//float feq0 = 0.1904761791f*rho+-0.597127747f*usqr                     
//float feq1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u    + 0.055555556f*(2.f*u*u-(v*v+w*w));
//float feq2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v    +-0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
//float feq3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u    + 0.055555556f*(2.f*u*u-(v*v+w*w));
//float feq4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v    +-0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
//float feq5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
//float feq6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
//float feq7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
//float feq8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
//float feq9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;   +-0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                           
//float feq10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
//float feq11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w)+-0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
//float feq12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)+ 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
//float feq13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w)+-0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
//float feq14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w    +-0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                           
//float feq15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w) + 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
//float feq16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w) +-0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
//float feq17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w) + 0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
//float feq18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w) +-0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;
//



//	float PI11 = f1 +f3 +f5 +f6 +f7 +f8 +f10+f12+f15+f17;
//	float PI22 = f2 +f4 +f5 +f6 +f7 +f8 +f11+f13+f16+f18;
//	float PI33 = f9 +f10+f11+f12+f13+f14+f15+f16+f17+f18;

}

__device__ void xsymmetry_bot(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0 && z == 0){
		f2 = f4;
        f13=f18;
        f11=f18;
        f16=f18;
		f6 =f7;
        f9 =f14;
        f12=f17;
	}
	else if(y == 0 && z == ZDIM-1){
		f4 = f2;
        f11=f13;
        f18=f13;
        f16=f13;
		f6 =f7;
        f14=f9;
        f17=f12;
	}
	else if(y == YDIM-1 && z == 0){
		f4 = f2;
        f11=f16;
        f18=f16;
        f13=f16;
		f7 =f6;
        f9 =f14;
        f12=f17;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4 = f2;
        f16=f11;
        f18=f11;
        f13=f11;
		f7 =f6;
        f14=f9;
        f17=f12;
	}
    else{
	if(y == 0){
        f2 = f4;
        f11=f13;
        f16=f18;
        f8 = f5;
    }
	else if(y == YDIM-1){
         f4=f2 ;
        f13=f11;
        f18=f16;
         f5=f8 ;
    }
//	if(z == 0){
//		f9  = f14;
//		f10 = f15;
//		f11 = f16;
//		f12 = f17;
//		f13 = f18;			
//	}    
//	else if(z == ZDIM-1){
//		f14 = f9;
//		f15 = f10;
//		f16 = f11;
//		f17 = f12;
//		f18 = f13;
//	}
    }
	f1 = f3 ;
	f5 = f6 ;
	f8 = f7 ;
	f10= f12;
	f15= f17;
}
__device__ void xsymmetry_top(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z)
{
	if(y == 0 && z == 0){
		f2  = f4;
		f13 = f18;			
		f11 = f18;
		f16 = f18;
		f5  = f8;
		f9  = f14;
		f10 = f15;
	}
	else if(y == 0 && z == ZDIM-1){
		f2  = f4;
		f11 = f13;
		f18 = f13;
		f16 = f13;
		f5  = f8;
		f14 = f9;
		f15 = f10;
	}
	else if(y == YDIM-1 && z == 0){
		f4  = f2;
		f18 = f16;
		f11 = f16;
		f13 = f16;	
		f8  = f5;
		f9  = f14;
		f10 = f15;
	}
	else if(y == YDIM-1 && z == ZDIM-1){
		f4  = f2;
		f13 = f11;
		f16 = f11;
		f18 = f11;
		f8  = f5;
		f14 = f9;
		f15 = f10;
	}

	else{
	if(y == 0){
		f2 = f4;
		f11 = f13;
		f16 = f18;
		f5 = f8;
	}
	else if(y == YDIM-1){
		f4 = f2;
		f13 = f11;
		f18 = f16;
		f8 = f5;
	}
//	else if(z == 0){
//		f9  = f14;
//		f10 = f15;
//		f11 = f16;
//		f12 = f17;
//		f13 = f18;			
//	}    
//	else if(z == ZDIM-1){
//		f14 = f9;
//		f15 = f10;
//		f16 = f11;
//		f17 = f12;
//		f18 = f13;
//	}
	}
	f3 = f1 ;
	f6 = f5 ;
	f7 = f8 ;
	f12= f10;
	f17= f15;
}

__device__ void ysymmetry_top(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int z)
{
	if(z == 0){
	f9 = f14;
	f10= f15;
	f11= f16;
	f12= f17;
	f13= f18;
	}
	if(z == ZDIM-1){
	f14= f9 ;
	f15= f10;
	f16= f11;
	f17= f12;
	f18= f13;
	}
	f4 = f2 ;
	f7 = f6 ;
	f8 = f5 ;
	f13= f11;
	f18= f16;
}

__device__ void ysymmetry_bot(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int z)
{
	if(z == 0){
	f9 = f14;
	f10= f15;
	f11= f16;
	f12= f17;
	f13= f18;
	}
	if(z == ZDIM-1){
	f14= f9 ;
	f15= f10;
	f16= f11;
	f17= f12;
	f18= f13;
	}
	f2 = f4 ;
	f6 = f7 ;
	f5 = f8 ;
	f11= f13;
	f16= f18;
}

__device__ void zsymmetry_top(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y)
{
	if(y == 0){
	f2 = f4 ;
	f6 = f7 ;
	f5 = f8 ;
	f11= f13;
	f16= f18;
	}
	if(y == YDIM-1){
	f4 = f2 ;
	f7 = f6 ;
	f8 = f5 ;
	f13= f11;
	f18= f16;
	}
	f14= f9 ;
	f15= f10;
	f16= f11;
	f17= f12;
	f18= f13;
}

__device__ void zsymmetry_bot(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y)
{
	if(y == 0){
	f2 = f4 ;
	f6 = f7 ;
	f5 = f8 ;
	f11= f13;
	f16= f18;
	}
	if(y == YDIM-1){
	f4 = f2 ;
	f7 = f6 ;
	f8 = f5 ;
	f13= f11;
	f18= f16;
	}
	f9 = f14;
	f10= f15;
	f11= f16;
	f12= f17;
	f13= f18;
}

inline __device__ void boundaries(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z, int im)
{
//	if(im == 3)//DirichletWest
//	{
//		DirichletWest(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
	if(im == 53)//DirichletWest
	{
		//DirichletWest_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
		DirichletWest_Regularized(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
	}
	else if(im == 54)//DirichletWest
	{
		//NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
		NeumannEast_Regularized(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
	}
//	if(im == 4)//DirichletWest
//	{
//		NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
//	if(im == 13)//DirichletWest
//	{
//		DirichletWest_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
//	else if(im == 14)//DirichletWest
//	{
//		NeumannEast_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
//	else if(im == 15)//DirichletNorth
//	{
//		DirichletNorth_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
//	if(im == 16)//DirichletSouth
//	{
//		DirichletSouth_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
	if(im == 21)//ysymm top
	{
		ysymmetry_top(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,z);
	}
	else if(im == 22)//ysymm bot
	{
		ysymmetry_bot(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,z);
	}
	else if(im == 23)//zsymm top
	{
		zsymmetry_top(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y);
	}
	else if(im == 24)//zsymm bot
	{
		zsymmetry_bot(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y);
	}
}
inline __device__ void boundaries_force(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, int y, int z, int im)
{
//	if(im == 3)//DirichletWest
//	{
//		DirichletWest(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
	if(im == 53)//DirichletWest
	{
		DirichletWest_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
		//DirichletWest_Regularized(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
	}
	else if(im == 54)//DirichletWest
	{
		NeumannEast(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
		//NeumannEast_Regularized(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
	}
//	else if(im == 15)//DirichletNorth
//	{
//		DirichletNorth_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
//	else if(im == 16)//DirichletSouth
//	{
//		DirichletSouth_Reg(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
//	}
	else if(im == 21)//ysymm top
	{
		ysymmetry_top(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,z);
	}
	else if(im == 22)//ysymm bot
	{
		ysymmetry_bot(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,z);
	}
	else if(im == 23)//zsymm top
	{
		zsymmetry_top(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y);
	}
	else if(im == 24)//zsymm bot
	{
		zsymmetry_bot(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y);
	}
	else if(im == 25)//zsymm top
	{
		xsymmetry_top(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
	}
	else if(im == 26)//zsymm bot
	{
		xsymmetry_bot(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z);
	}
}

inline __device__ void North_Extrap(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float rho)
{
	rho = 1.0f;
	float u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	float v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	float w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;

	float m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;

	m1  = -30.f*f0+-11.f*f1+-11.f*f2+-11.f*f3+-11.f*f4+  8.f*f5+  8.f*f6+  8.f*f7+  8.f*f8+-11.f*f9+  8.f*f10+  8.f*f11+  8.f*f12+  8.f*f13+-11.f*f14+  8.f*f15+  8.f*f16+  8.f*f17+  8.f*f18;
	m2  =  12.f*f0+ -4.f*f1+ -4.f*f2+ -4.f*f3+ -4.f*f4+    f5+    f6+    f7+    f8+ -4.f*f9+    f10+      f11+    f12+    f13+ -4.f*f14+    f15+    f16+    f17+    f18;
	m4  =           -4.f*f1         +  4.f*f3         +    f5+ -  f6+ -  f7+    f8         +    f10          + -  f12                    +    f15          + -  f17          ;
	m6  =                    -4.f*f2         +  4.f*f4+    f5+    f6+ -  f7+ -  f8                   +    f11          + -  f13                    +    f16          + -  f18;
	m8  =                                                                                 + -4.f*f9+    f10+    f11+    f12+    f13+  4.f*f14+ -  f15+ -  f16+ -  f17+ -  f18;
	m9  =            2.f*f1+ -  f2+  2.f*f3+ -  f4+    f5+    f6+    f7+    f8+ -  f9+    f10+ -2.f*f11+    f12+ -2.f*f13+ -  f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
	m10 =           -4.f*f1+  2.f*f2+ -4.f*f3+  2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+    f10+ -2.f*f11+    f12+ -2.f*f13+  2.f*f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
	m11 =                       f2         +    f4+    f5+    f6+    f7+    f8+ -  f9+ -  f10          + -  f12          + -  f14+ -  f15          + -  f17          ;
	m12 =                    -2.f*f2           -2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+ -  f10          + -  f12          +  2.f*f14+ -  f15          + -  f17          ;
	m13 =                                                  f5+ -  f6+    f7+ -  f8                                                                                                   ;
	m14 =                                                                                                         f11          + -  f13                    + -  f16          +    f18;
	m15 =                                                                                               f10          + -  f12                    + -  f15          +    f17          ;  
	m16 =                                                  f5+ -  f6+ -  f7+    f8           -  f10          +    f12                    + -  f15          +    f17          ;  
	m17 =                                               -  f5+ -  f6+    f7+    f8                   +    f11          + -  f13                    +    f16          + -  f18;  
	m18 =                                                                                               f10+ -  f11+    f12+ -  f13          + -  f15+    f16+ -  f17+    f18;

f0 =(0.052631579f*rho                           +- 0.012531328f*(m1)+ 0.047619048f*(m2));
f1 =(0.052631579f*rho+  0.1f*u                  +-0.0045948204f*(m1)+-0.015873016f*(m2)+  -0.1f*(m4)                 + 0.055555556f*((m9)-m10));
f2 =(0.052631579f*rho         +  0.1f*v         +-0.0045948204f*(m1)+-0.015873016f*(m2)             +   -0.1f*(m6)   +-0.027777778f*((m9)-m10)+ 0.083333333f*((m11)-m12));
f3 =(0.052631579f*rho+ -0.1f*u                  +-0.0045948204f*(m1)+-0.015873016f*(m2)+   0.1f*(m4)                 + 0.055555556f*((m9)-m10));                                                                                         
f4 =(0.052631579f*rho         + -0.1f*v         +-0.0045948204f*(m1)+-0.015873016f*(m2)             +    0.1f*(m6)   +-0.027777778f*((m9)-m10)+ 0.083333333f*((m11)-m12));
f5 =(0.052631579f*rho+  0.1f*u+  0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16-m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13))));
f6 =(0.052631579f*rho+ -0.1f*u+  0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16-m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13))));
f7 =(0.052631579f*rho+ -0.1f*u+ -0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16+m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13))));
f8 =(0.052631579f*rho+  0.1f*u+ -0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16+m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13))));
f9 =(0.052631579f*rho                  +  0.1f*w+-0.0045948204f*(m1)+-0.015873016f*(m2)                +   -0.1f*(m8)+-0.027777778f*((m9)-m10)+-0.083333333f*((m11)-m12));                                       
f10=(0.052631579f*rho+  0.1f*u         +  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16+m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15))));
f11=(0.052631579f*rho         +  0.1f*v+  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6+m8)+0.125f*( m17-m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +( 0.25f*(m14))));
f12=(0.052631579f*rho+ -0.1f*u         +  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16+m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15))));
f13=(0.052631579f*rho         + -0.1f*v+  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6-m8)+0.125f*(-m17-m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +(-0.25f*(m14))));
f14=(0.052631579f*rho                  + -0.1f*w+-0.0045948204f*(m1)+-0.015873016f*(m2)                +    0.1f*(m8)+-0.027777778f*((m9)-m10)+-0.083333333f*((m11)-m12));                                      
f15=(0.052631579f*rho+  0.1f*u         + -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16-m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15))));
f16=(0.052631579f*rho         +  0.1f*v+ -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6-m8)+0.125f*( m17+m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +(-0.25f*(m14))));
f17=(0.052631579f*rho+ -0.1f*u         + -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16-m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15))));
f18=(0.052631579f*rho         + -0.1f*v+ -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6+m8)+0.125f*(-m17+m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +( 0.25f*(m14))));

}

inline __device__ void South_Extrap(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float v)
{
	float rho,u,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = 0.f;//f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	w = 0.f;//f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;

	float m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;

	m1  = -30.f*f0+-11.f*f1+-11.f*f2+-11.f*f3+-11.f*f4+  8.f*f5+  8.f*f6+  8.f*f7+  8.f*f8+-11.f*f9+  8.f*f10+  8.f*f11+  8.f*f12+  8.f*f13+-11.f*f14+  8.f*f15+  8.f*f16+  8.f*f17+  8.f*f18;
	m2  =  12.f*f0+ -4.f*f1+ -4.f*f2+ -4.f*f3+ -4.f*f4+    f5+    f6+    f7+    f8+ -4.f*f9+    f10+      f11+    f12+    f13+ -4.f*f14+    f15+    f16+    f17+    f18;
	m4  =           -4.f*f1         +  4.f*f3         +    f5+ -  f6+ -  f7+    f8         +    f10          + -  f12                    +    f15          + -  f17          ;
	m6  =                    -4.f*f2         +  4.f*f4+    f5+    f6+ -  f7+ -  f8                   +    f11          + -  f13                    +    f16          + -  f18;
	m8  =                                                                                 + -4.f*f9+    f10+    f11+    f12+    f13+  4.f*f14+ -  f15+ -  f16+ -  f17+ -  f18;
	m9  =            2.f*f1+ -  f2+  2.f*f3+ -  f4+    f5+    f6+    f7+    f8+ -  f9+    f10+ -2.f*f11+    f12+ -2.f*f13+ -  f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
	m10 =           -4.f*f1+  2.f*f2+ -4.f*f3+  2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+    f10+ -2.f*f11+    f12+ -2.f*f13+  2.f*f14+    f15+ -2.f*f16+    f17+ -2.f*f18;
	m11 =                       f2         +    f4+    f5+    f6+    f7+    f8+ -  f9+ -  f10          + -  f12          + -  f14+ -  f15          + -  f17          ;
	m12 =                    -2.f*f2           -2.f*f4+    f5+    f6+    f7+    f8+  2.f*f9+ -  f10          + -  f12          +  2.f*f14+ -  f15          + -  f17          ;
	m13 =                                                  f5+ -  f6+    f7+ -  f8                                                                                                   ;
	m14 =                                                                                                         f11          + -  f13                    + -  f16          +    f18;
	m15 =                                                                                               f10          + -  f12                    + -  f15          +    f17          ;  
	m16 =                                                  f5+ -  f6+ -  f7+    f8           -  f10          +    f12                    + -  f15          +    f17          ;  
	m17 =                                               -  f5+ -  f6+    f7+    f8                   +    f11          + -  f13                    +    f16          + -  f18;  
	m18 =                                                                                               f10+ -  f11+    f12+ -  f13          + -  f15+    f16+ -  f17+    f18;

f0 =(0.052631579f*rho                           +- 0.012531328f*(m1)+ 0.047619048f*(m2));
f1 =(0.052631579f*rho+  0.1f*u                  +-0.0045948204f*(m1)+-0.015873016f*(m2)+  -0.1f*(m4)                 + 0.055555556f*((m9)-m10));
f2 =(0.052631579f*rho         +  0.1f*v         +-0.0045948204f*(m1)+-0.015873016f*(m2)             +   -0.1f*(m6)   +-0.027777778f*((m9)-m10)+ 0.083333333f*((m11)-m12));
f3 =(0.052631579f*rho+ -0.1f*u                  +-0.0045948204f*(m1)+-0.015873016f*(m2)+   0.1f*(m4)                 + 0.055555556f*((m9)-m10));                                                                                         
f4 =(0.052631579f*rho         + -0.1f*v         +-0.0045948204f*(m1)+-0.015873016f*(m2)             +    0.1f*(m6)   +-0.027777778f*((m9)-m10)+ 0.083333333f*((m11)-m12));
f5 =(0.052631579f*rho+  0.1f*u+  0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16-m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13))));
f6 =(0.052631579f*rho+ -0.1f*u+  0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16-m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13))));
f7 =(0.052631579f*rho+ -0.1f*u+ -0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16+m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13))));
f8 =(0.052631579f*rho+  0.1f*u+ -0.1f*v         + 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16+m17)+ (0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13))));
f9 =(0.052631579f*rho                  +  0.1f*w+-0.0045948204f*(m1)+-0.015873016f*(m2)                +   -0.1f*(m8)+-0.027777778f*((m9)-m10)+-0.083333333f*((m11)-m12));                                       
f10=(0.052631579f*rho+  0.1f*u         +  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16+m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15))));
f11=(0.052631579f*rho         +  0.1f*v+  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6+m8)+0.125f*( m17-m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +( 0.25f*(m14))));
f12=(0.052631579f*rho+ -0.1f*u         +  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16+m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15))));
f13=(0.052631579f*rho         + -0.1f*v+  0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6-m8)+0.125f*(-m17-m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +(-0.25f*(m14))));
f14=(0.052631579f*rho                  + -0.1f*w+-0.0045948204f*(m1)+-0.015873016f*(m2)                +    0.1f*(m8)+-0.027777778f*((m9)-m10)+-0.083333333f*((m11)-m12));                                      
f15=(0.052631579f*rho+  0.1f*u         + -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16-m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15))));
f16=(0.052631579f*rho         +  0.1f*v+ -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6-m8)+0.125f*( m17+m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +(-0.25f*(m14))));
f17=(0.052631579f*rho+ -0.1f*u         + -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16-m18)+ (0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15))));
f18=(0.052631579f*rho         + -0.1f*v+ -0.1f*w+ 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6+m8)+0.125f*(-m17+m18)-0.027777778f*(m10)+(-0.055555556f*(m9) +( 0.25f*(m14))));

}



int
timeval_subtract (double *result, struct timeval *x, struct timeval *y)
{
  struct timeval result0;

  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }

  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result0.tv_sec = x->tv_sec - y->tv_sec;
  result0.tv_usec = x->tv_usec - y->tv_usec;
  *result = ((double)result0.tv_usec)/1e6 + (double)result0.tv_sec;

  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}

inline __device__ void bgk_collide(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega)
{
	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9;
	rho +=f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float usqr = u*u+v*v+w*w;

//	f0 =(1.f-omega)*f0 +omega*(0.3333333333f*(rho-1.5f*usqr));
//	f1 =(1.f-omega)*f1 +omega*(0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
//	f2 =(1.f-omega)*f2 +omega*(0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
//	f3 =(1.f-omega)*f3 +omega*(0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
//	f4 =(1.f-omega)*f4 +omega*(0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
//	f5 =(1.f-omega)*f5 +omega*(0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr));
//	f6 =(1.f-omega)*f6 +omega*(0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
//	f7 =(1.f-omega)*f7 +omega*(0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
//	f8 =(1.f-omega)*f8 +omega*(0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr));
//	f9 =(1.f-omega)*f9 +omega*(0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr));
//	f10=(1.f-omega)*f10+omega*(0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr));
//	f11=(1.f-omega)*f11+omega*(0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(u+w)-1.5f*usqr));
//	f12=(1.f-omega)*f12+omega*(0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr));
//	f13=(1.f-omega)*f13+omega*(0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(u+w)-1.5f*usqr));
//	f14=(1.f-omega)*f14+omega*(0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr));
//	f15=(1.f-omega)*f15+omega*(0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr));
//	f16=(1.f-omega)*f16+omega*(0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr));
//	f17=(1.f-omega)*f17+omega*(0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr));
//	f18=(1.f-omega)*f18+omega*(0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr));

	f0 -=omega*(f0 -0.3333333333f*(rho-1.5f*usqr));
	f1 -=omega*(f1 -0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr));                
	f2 -=omega*(f2 -0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr));
	f3 -=omega*(f3 -0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr));
	f4 -=omega*(f4 -0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr));
	f5 -=omega*(f5 -0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr));
	f6 -=omega*(f6 -0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr));
	f7 -=omega*(f7 -0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr));
	f8 -=omega*(f8 -0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr));
	f9 -=omega*(f9 -0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr));
	f10-=omega*(f10-0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr));
	f11-=omega*(f11-0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(v+w)-1.5f*usqr));
	f12-=omega*(f12-0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr));
	f13-=omega*(f13-0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(-v+w)-1.5f*usqr));
	f14-=omega*(f14-0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr));
	f15-=omega*(f15-0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr));
	f16-=omega*(f16-0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr));
	f17-=omega*(f17-0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr));
	f18-=omega*(f18-0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr));

}

inline __device__ void mrt_collide(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega)
{
	float u,v,w;	
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
    float usqr = u*u+v*v+w*w;
//	u = rho*u;
//	v = rho*v;
//	w = rho*w;


	float m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;

	//COMPUTE M-MEQ
	//m1  = -19.f*f0+ 19.f*f5+19.f*f6+19.f*f7+19.f*f8+19.f*f10+19.f*f11+19.f*f12+19.f*f13+19.f*f15+19.f*f16+19.f*f17+19.f*f18   -19.f*(u*u+v*v+w*w);//+8.f*(f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18);
	//m4  = -3.33333333f*f1+3.33333333f*f3+1.66666667f*f5-1.66666667f*f6-1.66666667f*f7+1.66666667f*f8+1.66666667f*f10-1.66666667f*f12+1.66666667f*f15-1.66666667f*f17;
	//m6  = -3.33333333f*f2+3.33333333f*f4+1.66666667f*f5+1.66666667f*f6-1.66666667f*f7-1.66666667f*f8+1.66666667f*f11-1.66666667f*f13+1.66666667f*f16-1.66666667f*f18;
	//m8  = -3.33333333f*f9+1.66666667f*f10+1.66666667f*f11+1.66666667f*f12+1.66666667f*f13+3.33333333f*f14-1.66666667f*f15-1.66666667f*f16-1.66666667f*f17-1.66666667f*f18;
	m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));//+8.f*(f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18);
	m2  =  12.f*f0+ -4.f*f1+ -4.f*f2+ -4.f*f3+ -4.f*f4+      f5+      f6+      f7+      f8+ -4.f*f9+    f10+        f11+      f12+      f13+ -4.f*f14+      f15+      f16+      f17+      f18 +7.53968254f*(u*u+v*v+w*w);
//	m4  = 1.666666667f*(-2.f*f1+2.f*f3+f5-f6-f7+f8+f10-f12+f15-f17);
//	m6  = 1.666666667f*(-2.f*f2+2.f*f4+f5+f6-f7-f8+f11-f13+f16-f18);
//	m8  = 1.666666667f*(-2.f*f9+f10+f11+f12+f13+2.f*f14-f15-f16-f17-f18);
	m4  = 1.666666667f*(-3.f*f1+3.f*f3+u);
	m6  = 1.666666667f*(-3.f*f2+3.f*f4+v);
	m8  = 1.666666667f*(-3.f*f9+3.f*f14+w);
	m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
	m10 =-4.f*f1+ 2.f*f2+ -4.f*f3+ 2.f*f4+ f5+ f6+ f7+ f8+ 2.f*f9+ f10+ -2.f*f11+ f12+-2.f*f13+ 2.f*f14+ f15+ -2.f*f16+ f17+-2.f*f18;
	m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
	m12 =        -2.f*f2          -2.f*f4+ f5+ f6+ f7+ f8+ 2.f*f9+-f10          +-f12         + 2.f*f14+-f15          +-f17         ;
	m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
	m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
	m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;  
	m16 =                                  f5+-f6+-f7+ f8         -f10        +   f12                  +-f15          + f17         ;  
	m17 =                                 -f5+-f6+ f7+ f8              +     f11     +-    f13              +      f16     +-    f18;  
	m18 =                                                          f10+-     f11+ f12+-    f13         +-f15+      f16+-f17+     f18;

	if(SmagLES == "YES"){
////		float PI11 = -1.0f/38.0f*(     (m1)+19.0f*omega* (m9));
////		float PI22 = -1.0f/76.0f*(2.0f*(m1)-19.0f*(omega*(m9)-3.0f*omega*(m11)));
////		float PI33 = -1.0f/76.0f*(2.0f*(m1)-19.0f*(omega*(m9)+3.0f*omega*(m11)));
//		float PI11 = LRLEVEL*-0.026315789f*m1-0.5f *omega*m9;
//		float PI22 = LRLEVEL*-0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
//		float PI33 = LRLEVEL*-0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
//		float PI12 = LRLEVEL*-1.5f*omega*m13;
//		float PI23 = LRLEVEL*-1.5f*omega*m14;
//		float PI13 = LRLEVEL*-1.5f*omega*m15;
//		float nu0 = ((1.0f/omega)-0.5f)*LRFACTOR/3.0f;
//		float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));
//		//float Cs = 0.01f;
//		omega = 1.0f/(3.0f*(nu0+CS*Smag*LRFACTOR*LRFACTOR)*LRLEVEL+0.5f);
//		//omega = 1.0f/(1.0f/omega+3.f*CS*Smag*LRFACTOR*LRFACTOR);
//        //omega = 1.0f/(1.0f*LRLEVEL/1.99983f-1.f+0.5f+3.f*CS*Smag*LRFACTOR);

//float feq0 = 0.1904761791f*rho+-0.597127747f*usqr                                                 ;
float feq1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u                                                   ;
float feq2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v                       ;
float feq3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u                                                   ;
float feq4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v                       ;
float feq5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)                            ;
float feq6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)                            ;
float feq7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)                            ;
float feq8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)                            ;
float feq9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;
float feq10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)                                 ;
float feq11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w);
float feq12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)                            ;
float feq13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w);
float feq14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w;
float feq15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w)                            ;
float feq16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w);
float feq17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w)                            ;
float feq18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w);

feq1 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq2 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq3 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq4 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq5 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq6 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq7 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq8 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq9 += -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq10+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq11+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
feq12+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq13+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
feq14+= -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq15+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq16+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
feq17+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq18+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;

float PI11 = (f1-feq1)+(f3-feq3)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17);
float PI22 = (f2-feq2)+(f4-feq4)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
float PI33 = (f9-feq9)+(f14-feq14)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
float PI12 = (f5-feq5)+(f7-feq7)-(f6-feq6)-(f8-feq8);
float PI13 = (f10-feq10)+(f17-feq17)-(f12-feq12)-(f15-feq15);
float PI23 = (f11-feq11)+(f18-feq18)-(f13-feq13)-(f16-feq16);

float Q = sqrt(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13);
//float nu0 = ((1.0f/omega)-0.5f)*LRFACTOR/3.0f;
float tau0 = 1.f/omega;

//float Smag = (sqrt(nu0*nu0+18.f*CS*LRFACTOR*LRFACTOR*Q)-nu0)/(6.f*CS*LRFACTOR*LRFACTOR);
//float Smag = LRFACTOR*(sqrt(4.f/9.f*tau0*tau0+8.f*CS*LRFACTOR*Q)-2.f/3.f*tau0)/(4.f*CS*LRFACTOR*LRFACTOR);

//omega = 1.0f/(3.0f*(nu0+CS*Smag*LRFACTOR*LRFACTOR)*LRLEVEL+0.5f);

//float tau = tau0+0.5*(-tau0+sqrt(tau0*tau0+18.f*CS*LRFACTOR*Q));
float tau = tau0+0.5f*(-tau0+sqrt(tau0*tau0+18.f*CS*sqrt(2.f)*Q));
omega = 1.f/tau;

//float tau = 3.f*nu0*LRFACTOR+0.5f+(sqrt(tau0*tau0+18.f*CS*CS*LRFACTOR*LRFACTOR*Q)-tau0)*0.5f;
//omega = 1.f/tau;



	}


f0 -=- 0.012531328f*(m1)+ 0.047619048f*(m2);
f1 -=-0.0045948204f*(m1)+-0.015873016f*(m2)+  -0.1f*(m4)                 + 0.055555556f*((m9)*omega-m10);
f2 -=-0.0045948204f*(m1)+-0.015873016f*(m2)             +   -0.1f*(m6)   +-0.027777778f*((m9)*omega-m10)+ 0.083333333f*((m11)*omega-m12);
f3 -=-0.0045948204f*(m1)+-0.015873016f*(m2)+   0.1f*(m4)                 + 0.055555556f*((m9)*omega-m10);                                                                                         
f4 -=-0.0045948204f*(m1)+-0.015873016f*(m2)             +    0.1f*(m6)   +-0.027777778f*((m9)*omega-m10)+ 0.083333333f*((m11)*omega-m12);
f5 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16-m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13)));
f6 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16-m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13)));
f7 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16+m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13)));
f8 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16+m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13)));
f9 -=-0.0045948204f*(m1)+-0.015873016f*(m2)                +   -0.1f*(m8)+-0.027777778f*((m9)*omega-m10)+-0.083333333f*((m11)*omega-m12);                                       
f10-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16+m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15)));
f11-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6+m8)+0.125f*( m17-m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +( 0.25f*(m14)));
f12-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16+m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15)));
f13-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6-m8)+0.125f*(-m17-m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +(-0.25f*(m14)));
f14-=-0.0045948204f*(m1)+-0.015873016f*(m2)                +    0.1f*(m8)+-0.027777778f*((m9)*omega-m10)+-0.083333333f*((m11)*omega-m12);                                      
f15-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16-m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15)));
f16-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6-m8)+0.125f*( m17+m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +(-0.25f*(m14)));
f17-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16-m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15)));
f18-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6+m8)+0.125f*(-m17+m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +( 0.25f*(m14)));



}

inline __device__ void mrt_collide_LES(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega, float Cs)
{
	float u,v,w;	
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
    float usqr = u*u+v*v+w*w;
//	u = rho*u;
//	v = rho*v;
//	w = rho*w;


	float m1,m2,m4,m6,m8,m9,m10,m11,m12,m13,m14,m15,m16,m17,m18;

	//COMPUTE M-MEQ
	//m1  = -19.f*f0+ 19.f*f5+19.f*f6+19.f*f7+19.f*f8+19.f*f10+19.f*f11+19.f*f12+19.f*f13+19.f*f15+19.f*f16+19.f*f17+19.f*f18   -19.f*(u*u+v*v+w*w);//+8.f*(f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18);
	//m4  = -3.33333333f*f1+3.33333333f*f3+1.66666667f*f5-1.66666667f*f6-1.66666667f*f7+1.66666667f*f8+1.66666667f*f10-1.66666667f*f12+1.66666667f*f15-1.66666667f*f17;
	//m6  = -3.33333333f*f2+3.33333333f*f4+1.66666667f*f5+1.66666667f*f6-1.66666667f*f7-1.66666667f*f8+1.66666667f*f11-1.66666667f*f13+1.66666667f*f16-1.66666667f*f18;
	//m8  = -3.33333333f*f9+1.66666667f*f10+1.66666667f*f11+1.66666667f*f12+1.66666667f*f13+3.33333333f*f14-1.66666667f*f15-1.66666667f*f16-1.66666667f*f17-1.66666667f*f18;
	m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));//+8.f*(f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18);
	m2  =  12.f*f0+ -4.f*f1+ -4.f*f2+ -4.f*f3+ -4.f*f4+      f5+      f6+      f7+      f8+ -4.f*f9+    f10+        f11+      f12+      f13+ -4.f*f14+      f15+      f16+      f17+      f18 +7.53968254f*(u*u+v*v+w*w);
//	m4  = 1.666666667f*(-2.f*f1+2.f*f3+f5-f6-f7+f8+f10-f12+f15-f17);
//	m6  = 1.666666667f*(-2.f*f2+2.f*f4+f5+f6-f7-f8+f11-f13+f16-f18);
//	m8  = 1.666666667f*(-2.f*f9+f10+f11+f12+f13+2.f*f14-f15-f16-f17-f18);
	m4  = 1.666666667f*(-3.f*f1+3.f*f3+u);
	m6  = 1.666666667f*(-3.f*f2+3.f*f4+v);
	m8  = 1.666666667f*(-3.f*f9+3.f*f14+w);
	m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
	m10 =-4.f*f1+ 2.f*f2+ -4.f*f3+ 2.f*f4+ f5+ f6+ f7+ f8+ 2.f*f9+ f10+ -2.f*f11+ f12+-2.f*f13+ 2.f*f14+ f15+ -2.f*f16+ f17+-2.f*f18;
	m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
	m12 =        -2.f*f2          -2.f*f4+ f5+ f6+ f7+ f8+ 2.f*f9+-f10          +-f12         + 2.f*f14+-f15          +-f17         ;
	m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
	m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
	m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;  
	m16 =                                  f5+-f6+-f7+ f8         -f10        +   f12                  +-f15          + f17         ;  
	m17 =                                 -f5+-f6+ f7+ f8              +     f11     +-    f13              +      f16     +-    f18;  
	m18 =                                                          f10+-     f11+ f12+-    f13         +-f15+      f16+-f17+     f18;

	if(SmagLES == "YES"){
//		float PI11 = -0.026315789f*m1-0.5f *omega*m9;
//		float PI22 = -0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
//		float PI33 = -0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
//
//		float PI12 = -1.5f*omega*m13;
//		float PI23 = -1.5f*omega*m14;
//		float PI13 = -1.5f*omega*m15;
//		float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));
//		omega = 1.0f/(1.0f/omega+3.f*CS*Smag);

//		float PI11 = LRLEVEL*-1.0f/38.0f*(     (m1)+19.0f*omega* (m9));
//		float PI22 = LRLEVEL*-1.0f/76.0f*(2.0f*(m1)-19.0f*(omega*(m9)-3.0f*omega*(m11)));
//		float PI33 = LRLEVEL*-1.0f/76.0f*(2.0f*(m1)-19.0f*(omega*(m9)+3.0f*omega*(m11)));
//		float PI12 = LRLEVEL*-1.5f*omega*m13;
//		float PI23 = LRLEVEL*-1.5f*omega*m14;
//		float PI13 = LRLEVEL*-1.5f*omega*m15;
//		float nu0 = ((1.0f/omega)-0.5f)/3.0f;
//		float Smag = sqrt(PI11*PI11+PI22*PI22+PI33*PI33+PI12*PI12+PI23*PI23+PI13*PI13);
//		omega = 1.0f/(3.0f*(nu0+Cs*Smag*LRLEVEL*LRLEVEL)+0.5f);


//float feq0 = 0.1904761791f*rho+-0.597127747f*usqr                                                 ;
float feq1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u                                                   ;
float feq2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v                       ;
float feq3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u                                                   ;
float feq4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v                       ;
float feq5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)                            ;
float feq6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)                            ;
float feq7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)                            ;
float feq8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)                            ;
float feq9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;
float feq10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)                                 ;
float feq11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w);
float feq12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)                            ;
float feq13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w);
float feq14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w;
float feq15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w)                            ;
float feq16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w);
float feq17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w)                            ;
float feq18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w);

feq1 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq2 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq3 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq4 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq5 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq6 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq7 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq8 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq9 += -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq10+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq11+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
feq12+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq13+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
feq14+= -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq15+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq16+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
feq17+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq18+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;

float PI11 = (f1-feq1)+(f3-feq3)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17);
float PI22 = (f2-feq2)+(f4-feq4)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
float PI33 = (f9-feq9)+(f14-feq14)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
float PI12 = (f5-feq5)+(f7-feq7)-(f6-feq6)-(f8-feq8);
float PI13 = (f10-feq10)+(f17-feq17)-(f12-feq12)-(f15-feq15);
float PI23 = (f11-feq11)+(f18-feq18)-(f13-feq13)-(f16-feq16);

//float Q = sqrt(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13);
//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
//
//float Smag = (sqrt(nu0*nu0+18.f*CS*Q)-nu0)/(6.f*CS);
//
////omega = 1.0f/(1.0f/omega+3.f*CS*Smag);
//
//float tau0 = 1.f/omega;
//float tau = 3.f*nu0+0.5f+(sqrt(tau0*tau0+18.f*CS*CS*Q)-tau0)*0.5f;
//omega = 1.f/tau;

float Q = sqrt(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13);
//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
float tau0 = 1.f/omega;

//float Smag = (sqrt(nu0*nu0+18.f*CS*LRFACTOR*LRFACTOR*Q)-nu0)/(6.f*CS*LRFACTOR*LRFACTOR);
//float Smag = (sqrt(4.f/9.f*tau0*tau0+8.f*CS*Q)-2.f/3.f*tau0)/(4.f*CS);

//omega = 1.0f/(3.0f*(nu0+CS*Smag)+0.5f);

float tau = tau0+0.5f*(-tau0+sqrt(tau0*tau0+18.f*sqrt(2.f)*CS*Q));
omega = 1.f/tau;




	}


f0 -=- 0.012531328f*(m1)+ 0.047619048f*(m2);
f1 -=-0.0045948204f*(m1)+-0.015873016f*(m2)+  -0.1f*(m4)                 + 0.055555556f*((m9)*omega-m10);
f2 -=-0.0045948204f*(m1)+-0.015873016f*(m2)             +   -0.1f*(m6)   +-0.027777778f*((m9)*omega-m10)+ 0.083333333f*((m11)*omega-m12);
f3 -=-0.0045948204f*(m1)+-0.015873016f*(m2)+   0.1f*(m4)                 + 0.055555556f*((m9)*omega-m10);                                                                                         
f4 -=-0.0045948204f*(m1)+-0.015873016f*(m2)             +    0.1f*(m6)   +-0.027777778f*((m9)*omega-m10)+ 0.083333333f*((m11)*omega-m12);
f5 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16-m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13)));
f6 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16-m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13)));
f7 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*(-m16+m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+( 0.25f*(m13)));
f8 -= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m6)              +0.013888889f*(m10)+0.041666667f*(m12)+0.125f*( m16+m17)+ omega*(0.027777778f*(m9) +0.083333333f*(m11)+(-0.25f*(m13)));
f9 -=-0.0045948204f*(m1)+-0.015873016f*(m2)                +   -0.1f*(m8)+-0.027777778f*((m9)*omega-m10)+-0.083333333f*((m11)*omega-m12);                                       
f10-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16+m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15)));
f11-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6+m8)+0.125f*( m17-m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +( 0.25f*(m14)));
f12-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16+m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15)));
f13-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6-m8)+0.125f*(-m17-m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +(-0.25f*(m14)));
f14-=-0.0045948204f*(m1)+-0.015873016f*(m2)                +    0.1f*(m8)+-0.027777778f*((m9)*omega-m10)+-0.083333333f*((m11)*omega-m12);                                      
f15-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+ 0.025f*(m4-m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*(-m16-m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+(-0.25f*(m15)));
f16-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             +  0.025f*(m6-m8)+0.125f*( m17+m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +(-0.25f*(m14)));
f17-= 0.0033416876f*(m1)+ 0.003968254f*(m2)+-0.025f*(m4+m8)              +0.013888889f*(m10)-0.041666667f*(m12)+0.125f*( m16-m18)+ omega*(0.027777778f*(m9) -0.083333333f*(m11)+( 0.25f*(m15)));
f18-= 0.0033416876f*(m1)+ 0.003968254f*(m2)             + -0.025f*(m6+m8)+0.125f*(-m17+m18)-0.027777778f*(m10)+omega*(-0.055555556f*(m9) +( 0.25f*(m14)));



}

inline __device__ void bgk_scale_cf(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float SF)
{
	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float usqr = u*u+v*v+w*w;

    float feq0  = 0.3333333333f*(rho-1.5f*usqr);
    float feq1  = 0.0555555556f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
    float feq2  = 0.0555555556f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
    float feq3  = 0.0555555556f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
    float feq4  = 0.0555555556f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
    float feq5  = 0.0277777778f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
    float feq6  = 0.0277777778f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
    float feq7  = 0.0277777778f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
    float feq8  = 0.0277777778f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
    float feq9  = 0.0555555556f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
    float feq10 = 0.0277777778f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
    float feq11 = 0.0277777778f*(rho+3.0f*(v+w)+4.5f*(v+w)*(v+w)-1.5f*usqr);
    float feq12 = 0.0277777778f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
    float feq13 = 0.0277777778f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(-v+w)-1.5f*usqr);
    float feq14 = 0.0555555556f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
    float feq15 = 0.0277777778f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
    float feq16 = 0.0277777778f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
    float feq17 = 0.0277777778f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
    float feq18 = 0.0277777778f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);

    f0 =SF*f0 +(1.0f-SF)*feq0 ;
    f1 =SF*f1 +(1.0f-SF)*feq1 ;
    f2 =SF*f2 +(1.0f-SF)*feq2 ;
    f3 =SF*f3 +(1.0f-SF)*feq3 ;
    f4 =SF*f4 +(1.0f-SF)*feq4 ;
    f5 =SF*f5 +(1.0f-SF)*feq5 ;
    f6 =SF*f6 +(1.0f-SF)*feq6 ;
    f7 =SF*f7 +(1.0f-SF)*feq7 ;
    f8 =SF*f8 +(1.0f-SF)*feq8 ;
    f9 =SF*f9 +(1.0f-SF)*feq9 ;
    f10=SF*f10+(1.0f-SF)*feq10;
    f11=SF*f11+(1.0f-SF)*feq11;
    f12=SF*f12+(1.0f-SF)*feq12;
    f13=SF*f13+(1.0f-SF)*feq13;
    f14=SF*f14+(1.0f-SF)*feq14;
    f15=SF*f15+(1.0f-SF)*feq15;
    f16=SF*f16+(1.0f-SF)*feq16;
    f17=SF*f17+(1.0f-SF)*feq17;
    f18=SF*f18+(1.0f-SF)*feq18;

}

inline __device__ void mrt_scale_cf(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float SF)
{
	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float usqr = u*u+v*v+w*w;
                                                                                                                
float feq0 = 0.1904761791f*rho+-0.597127747f*usqr                                                 ;
float feq1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u                                                   ;
float feq2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v                       ;
float feq3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u                                                   ;
float feq4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v                       ;
float feq5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)                            ;
float feq6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)                            ;
float feq7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)                            ;
float feq8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)                            ;
float feq9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;
float feq10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)                                 ;
float feq11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w);
float feq12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)                            ;
float feq13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w);
float feq14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w;
float feq15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w)                            ;
float feq16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w);
float feq17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w)                            ;
float feq18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w);

feq1 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq2 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq3 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq4 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq5 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq6 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq7 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq8 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq9 += -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq10+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq11+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
feq12+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq13+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
feq14+= -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq15+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq16+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
feq17+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq18+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;

//float m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18)   -19.f*(u*u+v*v+w*w);
//float m2  = 12.f*f0+-4.f*f1+-4.f*f2+-4.f*f3+-4.f*f4+f5+f6+f7+f8+-4.f*f9+f10+f11+f12+f13+-4.f*f14+f15+f16+f17+f18 +7.53968254f*(u*u+v*v+w*w);
//float m4  = 1.666666667f*(-2.f*f1+2.f*f3+f5-f6-f7+f8+f10-f12+f15-f17);
//float m6  = 1.666666667f*(-2.f*f2+2.f*f4+f5+f6-f7-f8+f11-f13+f16-f18);
//float m8  = 1.666666667f*(-2.f*f9+f10+f11+f12+f13+2.f*f14-f15-f16-f17-f18);
//float m4  = 1.666666667f*(-3.f*f1+3.f*f3+u);
//float m6  = 1.666666667f*(-3.f*f2+3.f*f4+v);
//float m8  = 1.666666667f*(-3.f*f9+3.f*f14+w);
//float m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
//float m10 =-4.f*f1+ 2.f*f2+ -4.f*f3+ 2.f*f4+ f5+ f6+ f7+ f8+ 2.f*f9+ f10+ -2.f*f11+ f12+-2.f*f13+ 2.f*f14+ f15+ -2.f*f16+ f17+-2.f*f18;
//float m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
//float m12 =        -2.f*f2          -2.f*f4+ f5+ f6+ f7+ f8+ 2.f*f9+-f10          +-f12         + 2.f*f14+-f15          +-f17         ;
//float m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
//float m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
//float m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;  
//float m16 =                                  f5+-f6+-f7+ f8         -f10        +   f12                  +-f15          + f17         ;  
//float m17 =                                 -f5+-f6+ f7+ f8              +     f11     +-    f13              +      f16     +-    f18;  
//float m18 =                                                          f10+-     f11+ f12+-    f13         +-f15+      f16+-f17+     f18;





float m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));
float m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
float m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
float m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
float m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
float m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;

float omega = 1.0f/(3.0f*(UMAX*OBSTR1*2.f/RE)+0.5f);
float omega2 = 2.0f/(1.0f+2.0f*(2.0f/omega-1.0f));

float PI11 = -0.026315789f*m1-0.5f *omega*m9;
float PI22 = -0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
float PI33 = -0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
float PI12 = -1.5f*omega*m13;
float PI23 = -1.5f*omega*m14;
float PI13 = -1.5f*omega*m15;
//we know Smag on coarse mesh
float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));
//omega = 1.0f/(3.0f*(nu0+Cs*Smag*sqrt(2.f))+0.5f);
//omega  = 1.0f/(1.0f/omega+3.f*CS*Smag);
//omega2 = 1.0f/(1.0f/omega2+3.f*CS*Smag*sqrt(2.f)*LRFACTOR*LRFACTOR);
//omega  = 1.0f/(1.0f/omega +3.f*CS*Smag);
//omega2 = 1.0f/(1.0f/omega2+3.f*CS*Smag*sqrt(2.f)*LRFACTOR*LRFACTOR);
//omega  = 1.0f/(1.0f/omega +3.f*CS*Smag);
//omega2 = 1.0f/(1.0f*LRLEVEL/omega2-1.f+0.5f+3.f*CS*Smag*sqrt(2.f)*LRFACTOR);

//float PI11 = (f1-feq1)+(f3-feq3)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17);
//float PI22 = (f2-feq2)+(f4-feq4)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
//float PI33 = (f9-feq9)+(f14-feq14)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
//float PI12 = (f5-feq5)+(f7-feq7)-(f6-feq6)-(f8-feq8);
//float PI13 = (f10-feq10)+(f17-feq17)-(f12-feq12)-(f15-feq15);
//float PI23 = (f11-feq11)+(f18-feq18)-(f13-feq13)-(f16-feq16);
//
//float Q = sqrt(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13);
//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
//float tau0c = 1.f/omega;
//float tau = tau0c+0.5*(-tau0c+sqrt(tau0c*tau0c+18.f*CS*Q));//tau_total of coarse mesh
//omega = 1.f/tau;//total omega on coarse mesh
//tau = tau0+0.5*(-tau0+sqrt(tau0*tau0+18.f*CS*LRFACTOR*Q));
//omega2= 1.f/tau;

SF = (omega*(1.0f-omega2))/((1.0f-omega)*omega2);//for post-collision 
//SF = omega*0.5f/omega2;//for post-streaming, pre-collision?





f0 =SF*f0 +(1.0f-SF)*feq0 ;
f1 =SF*f1 +(1.0f-SF)*feq1 ;
f2 =SF*f2 +(1.0f-SF)*feq2 ;
f3 =SF*f3 +(1.0f-SF)*feq3 ;
f4 =SF*f4 +(1.0f-SF)*feq4 ;
f5 =SF*f5 +(1.0f-SF)*feq5 ;
f6 =SF*f6 +(1.0f-SF)*feq6 ;
f7 =SF*f7 +(1.0f-SF)*feq7 ;
f8 =SF*f8 +(1.0f-SF)*feq8 ;
f9 =SF*f9 +(1.0f-SF)*feq9 ;
f10=SF*f10+(1.0f-SF)*feq10;
f11=SF*f11+(1.0f-SF)*feq11;
f12=SF*f12+(1.0f-SF)*feq12;
f13=SF*f13+(1.0f-SF)*feq13;
f14=SF*f14+(1.0f-SF)*feq14;
f15=SF*f15+(1.0f-SF)*feq15;
f16=SF*f16+(1.0f-SF)*feq16;
f17=SF*f17+(1.0f-SF)*feq17;
f18=SF*f18+(1.0f-SF)*feq18;

}
inline __device__ void mrt_scale_fc_LES(float& f0, float& f1, float& f2,
					float& f3 , float& f4 , float& f5 ,
					float& f6 , float& f7 , float& f8 , float& f9,
					float& f10, float& f11, float& f12,
					float& f13, float& f14, float& f15,
					float& f16, float& f17, float& f18, float omega, float omega2)
{
	float rho,u,v,w;	
	rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9+
	      f10+f11+f12+f13+f14+f15+f16+f17+f18;
	u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
	v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
	w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	float usqr = u*u+v*v+w*w;
                                                                                                                
float feq0 = 0.1904761791f*rho+-0.597127747f*usqr                                                 ;
float feq1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u                                                   ;
float feq2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v                       ;
float feq3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u                                                   ;
float feq4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v                       ;
float feq5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)                            ;
float feq6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)                            ;
float feq7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)                            ;
float feq8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)                            ;
float feq9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;
float feq10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)                                 ;
float feq11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w);
float feq12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)                            ;
float feq13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w);
float feq14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w;
float feq15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w)                            ;
float feq16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w);
float feq17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w)                            ;
float feq18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w);

feq1 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq2 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq3 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
feq4 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
feq5 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq6 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq7 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
feq8 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
feq9 += -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq10+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq11+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
feq12+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq13+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
feq14+= -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
feq15+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
feq16+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
feq17+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
feq18+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;

//float m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));
//float m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
//float m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
//float m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
//float m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
//float m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;

//float PI11 = -0.026315789f*m1-0.5f *omega*m9;
//float PI22 = -0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
//float PI33 = -0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
//float PI12 = -1.5f*omega*m13;
//float PI23 = -1.5f*omega*m14;
//float PI13 = -1.5f*omega*m15;
////we know Smag on fine mesh. Smag_c=Smag_f*sqrt(2)
//float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));
//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
////omega = 1.0f/(3.0f*(nu0+CS*Smag*sqrt(2.f))+0.5f);
////omega2 = 1.0f/(1.0f/omega2+3.f*CS*Smag*LRFACTOR);
////omega  = 1.0f/(1.0f/omega+3.f*CS*Smag/sqrt(2.f));
////omega2 = 1.0f/(1.0f*LRLEVEL/omega2-1.f+0.5f+3.f*CS*Smag*LRFACTOR);
////omega  = 1.0f/(1.0f/omega+3.f*CS*Smag/sqrt(2.f));

//float PI11 = (f1-feq1)+(f3-feq3)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17);
//float PI22 = (f2-feq2)+(f4-feq4)+(f5-feq5)+(f6-feq6)+(f7-feq7)+(f8-feq8)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
//float PI33 = (f9-feq9)+(f14-feq14)+(f10-feq10)+(f12-feq12)+(f15-feq15)+(f17-feq17)+(f11-feq11)+(f13-feq13)+(f16-feq16)+(f18-feq18);
//float PI12 = (f5-feq5)+(f7-feq7)-(f6-feq6)-(f8-feq8);
//float PI13 = (f10-feq10)+(f17-feq17)-(f12-feq12)-(f15-feq15);
//float PI23 = (f11-feq11)+(f18-feq18)-(f13-feq13)-(f16-feq16);
//
//float Q = sqrt(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13);
//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
//float tau0f = 1.f/omega2;
//float tau0c = 1.f/omega;
//float tau = tau0f+0.5*(-tau0f+sqrt(tau0f*tau0f+18.f*CS*sqrt(2.f)*Q));//tau_total of fine
//omega2 = 1.f/tau;//total omega on fine mesh
//tau = LRLEVEL*(tau-tau0f)+tau0c;
//omega= 1.f/tau;

//tau = tau0+0.5*(-tau0+sqrt(tau0*tau0+18.f*CS*Q));

float SF = (omega*(1.0f-omega2))/((1.0f-omega)*omega2);
//float SF = omega2*2.f/omega;



//float SF = ((1.0f-omega)*omega2/LRFACTOR)/(omega*(1.0f-omega2));
//SF = omega*2.f/omega2;

f0 =SF*f0 +(1.0f-SF)*feq0 ;
f1 =SF*f1 +(1.0f-SF)*feq1 ;
f2 =SF*f2 +(1.0f-SF)*feq2 ;
f3 =SF*f3 +(1.0f-SF)*feq3 ;
f4 =SF*f4 +(1.0f-SF)*feq4 ;
f5 =SF*f5 +(1.0f-SF)*feq5 ;
f6 =SF*f6 +(1.0f-SF)*feq6 ;
f7 =SF*f7 +(1.0f-SF)*feq7 ;
f8 =SF*f8 +(1.0f-SF)*feq8 ;
f9 =SF*f9 +(1.0f-SF)*feq9 ;
f10=SF*f10+(1.0f-SF)*feq10;
f11=SF*f11+(1.0f-SF)*feq11;
f12=SF*f12+(1.0f-SF)*feq12;
f13=SF*f13+(1.0f-SF)*feq13;
f14=SF*f14+(1.0f-SF)*feq14;
f15=SF*f15+(1.0f-SF)*feq15;
f16=SF*f16+(1.0f-SF)*feq16;
f17=SF*f17+(1.0f-SF)*feq17;
f18=SF*f18+(1.0f-SF)*feq18;

}

__device__ int dmin(int a, int b)
{
	if (a<b) return a;
	else return b-1;
}
__device__ int dmax(int a)
{
	if (a>-1) return a;
	else return 0;
}
__device__ int dmin_p(int a, int b)
{
	if (a<b) return a;
	else return 0;
}
__device__ int dmax_p(int a, int b)
{
	if (a>-1) return a;
	else return b-1;
}


inline __device__ int f_mem(int f_num, int x, int y, int z, size_t pitch, int zInner)
{
	int index = (x+y*pitch+z*YDIM*pitch)+f_num*pitch*YDIM*(zInner);
	index = dmax(index);
	index = dmin(index,19*pitch*YDIM*(zInner));
//	if(index<0) index = 0;
//	else if(index>19*pitch*YDIM*ZDIM/GPU_N-2) index = 19*pitch*(YDIM*ZDIM/GPU_N-2);
	return index;
}

inline __device__ int buff_mem(int f_num, int x, int y, size_t pitch)
{
	int index = (x+y*pitch)+f_num*pitch*YDIM;
	index = dmax(index);
	index = dmin(index,19*pitch*YDIM);
//	if(index<0) index = 0;
//	else if(index>19*pitch*YDIM) index = 19*pitch*YDIM;
	return index;
}

__global__ void update_inner(float* fA, float* fB, float* g, float* h,
							float omega, size_t pitch, int GPU, int zInner)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y,GPU*(zInner+2)+1+z);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;

//	if(REFINEMENT == "YES" && x > LRX0+1 && x < LRX0+(XLRDIM-1)*LRFACTOR-1 
//		&& y > LRY0+1 && y < LRY0+(YLRDIM-1)*LRFACTOR-1 && z > LRZ0+1 && z < LRZ0+(ZLRDIM-1)*LRFACTOR-1 ||
//		(x>XDIM-1)){
//	}
//	else{
    f0 = fA[j];
	f1 = fA[f_mem   (1 ,x-1,y  ,z  ,pitch, zInner)];
	f3 = fA[f_mem   (3 ,x+1,y  ,z  ,pitch, zInner)];
	f2 = fA[f_mem   (2 ,x  ,y-1,z  ,pitch, zInner)];
	f5 = fA[f_mem   (5 ,x-1,y-1,z  ,pitch, zInner)];
	f6 = fA[f_mem   (6 ,x+1,y-1,z  ,pitch, zInner)];
	f4 = fA[f_mem   (4 ,x  ,y+1,z  ,pitch, zInner)];
	f7 = fA[f_mem   (7 ,x+1,y+1,z  ,pitch, zInner)];
	f8 = fA[f_mem   (8 ,x-1,y+1,z  ,pitch, zInner)];

    if(z==zInner){//top nodes need info from h
	f9 = fA[f_mem   (9 ,x  ,y  ,z-1,pitch, zInner)];
	f10= fA[f_mem   (10,x-1,y  ,z-1,pitch, zInner)];
	f11= fA[f_mem   (11,x  ,y-1,z-1,pitch, zInner)];
	f12= fA[f_mem   (12,x+1,y  ,z-1,pitch, zInner)];
	f13= fA[f_mem   (13,x  ,y+1,z-1,pitch, zInner)];
	f14= h [buff_mem(14,x  ,y  ,pitch)];
	f15= h [buff_mem(15,x-1,y  ,pitch)];
	f16= h [buff_mem(16,x  ,y-1,pitch)];
	f17= h [buff_mem(17,x+1,y  ,pitch)];
	f18= h [buff_mem(18,x  ,y+1,pitch)];
    }
    else if(z==0){//bottom nodes need info from g
	f9 = g [buff_mem(9 ,x  ,y  ,pitch)];
	f10= g [buff_mem(10,x-1,y  ,pitch)];
	f11= g [buff_mem(11,x  ,y-1,pitch)];
	f12= g [buff_mem(12,x+1,y  ,pitch)];
	f13= g [buff_mem(13,x  ,y+1,pitch)];
	f14= fA[f_mem   (14,x  ,y  ,z+1,pitch, zInner)];
	f15= fA[f_mem   (15,x-1,y  ,z+1,pitch, zInner)];
	f16= fA[f_mem   (16,x  ,y-1,z+1,pitch, zInner)];
	f17= fA[f_mem   (17,x+1,y  ,z+1,pitch, zInner)];
	f18= fA[f_mem   (18,x  ,y+1,z+1,pitch, zInner)];
    }
    else{//normal nodes
	f9 = fA[f_mem(9 ,x  ,y  ,z,pitch,zInner)];
	f10= fA[f_mem(10,x-1,y  ,z,pitch,zInner)];
	f11= fA[f_mem(11,x  ,y-1,z,pitch,zInner)];
	f12= fA[f_mem(12,x+1,y  ,z,pitch,zInner)];
	f13= fA[f_mem(13,x  ,y+1,z,pitch,zInner)];
	f14= fA[f_mem(14,x  ,y  ,z,pitch,zInner)];
	f15= fA[f_mem(15,x-1,y  ,z,pitch,zInner)];
	f16= fA[f_mem(16,x  ,y-1,z,pitch,zInner)];
	f17= fA[f_mem(17,x+1,y  ,z,pitch,zInner)];
	f18= fA[f_mem(18,x  ,y+1,z,pitch,zInner)];

    }//end normal nodes

	if(im == 1 || im ==10){//BB
		fB[f_mem(1 ,x,y,z,pitch,zInner)] = f3 ;
		fB[f_mem(2 ,x,y,z,pitch,zInner)] = f4 ;
		fB[f_mem(3 ,x,y,z,pitch,zInner)] = f1 ;
		fB[f_mem(4 ,x,y,z,pitch,zInner)] = f2 ;
		fB[f_mem(5 ,x,y,z,pitch,zInner)] = f7 ;
		fB[f_mem(6 ,x,y,z,pitch,zInner)] = f8 ;
		fB[f_mem(7 ,x,y,z,pitch,zInner)] = f5 ;
		fB[f_mem(8 ,x,y,z,pitch,zInner)] = f6 ;
		fB[f_mem(9 ,x,y,z,pitch,zInner)] = f14;
		fB[f_mem(10,x,y,z,pitch,zInner)] = f17;
		fB[f_mem(11,x,y,z,pitch,zInner)] = f18;
		fB[f_mem(12,x,y,z,pitch,zInner)] = f15;
		fB[f_mem(13,x,y,z,pitch,zInner)] = f16;
		fB[f_mem(14,x,y,z,pitch,zInner)] = f9 ;
		fB[f_mem(15,x,y,z,pitch,zInner)] = f12;
		fB[f_mem(16,x,y,z,pitch,zInner)] = f13;
		fB[f_mem(17,x,y,z,pitch,zInner)] = f10;
		fB[f_mem(18,x,y,z,pitch,zInner)] = f11;
	}
	else{

        if(im == 100)//north outlet
        {
           	f0 = fA[f_mem(0 ,x,y-1,z,pitch,zInner)];
        	f1 = fA[f_mem(1 ,x,y-1,z,pitch,zInner)];
        	f3 = fA[f_mem(3 ,x,y-1,z,pitch,zInner)];
        	f2 = fA[f_mem(2 ,x,y-1,z,pitch,zInner)];
        	f5 = fA[f_mem(5 ,x,y-1,z,pitch,zInner)];
        	f6 = fA[f_mem(6 ,x,y-1,z,pitch,zInner)];
        	f4 = fA[f_mem(4 ,x,y-1,z,pitch,zInner)];
        	f7 = fA[f_mem(7 ,x,y-1,z,pitch,zInner)];
        	f8 = fA[f_mem(8 ,x,y-1,z,pitch,zInner)];
        	f9 = fA[f_mem(9 ,x,y-1,z,pitch,zInner)];
        	f10= fA[f_mem(10,x,y-1,z,pitch,zInner)];
        	f11= fA[f_mem(11,x,y-1,z,pitch,zInner)];
        	f12= fA[f_mem(12,x,y-1,z,pitch,zInner)];
        	f13= fA[f_mem(13,x,y-1,z,pitch,zInner)];
        	f14= fA[f_mem(14,x,y-1,z,pitch,zInner)];
        	f15= fA[f_mem(15,x,y-1,z,pitch,zInner)];
        	f16= fA[f_mem(16,x,y-1,z,pitch,zInner)];
        	f17= fA[f_mem(17,x,y-1,z,pitch,zInner)];
        	f18= fA[f_mem(18,x,y-1,z,pitch,zInner)];

			North_Extrap(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,1.0f);
        }
        if(im == 200)//south inlet
        {
           	f0 = fA[f_mem(0 ,x,y+1,z,pitch,zInner)];
        	f1 = fA[f_mem(1 ,x,y+1,z,pitch,zInner)];
        	f3 = fA[f_mem(3 ,x,y+1,z,pitch,zInner)];
        	f2 = fA[f_mem(2 ,x,y+1,z,pitch,zInner)];
        	f5 = fA[f_mem(5 ,x,y+1,z,pitch,zInner)];
        	f6 = fA[f_mem(6 ,x,y+1,z,pitch,zInner)];
        	f4 = fA[f_mem(4 ,x,y+1,z,pitch,zInner)];
        	f7 = fA[f_mem(7 ,x,y+1,z,pitch,zInner)];
        	f8 = fA[f_mem(8 ,x,y+1,z,pitch,zInner)];
        	f9 = fA[f_mem(9 ,x,y+1,z,pitch,zInner)];
        	f10= fA[f_mem(10,x,y+1,z,pitch,zInner)];
        	f11= fA[f_mem(11,x,y+1,z,pitch,zInner)];
        	f12= fA[f_mem(12,x,y+1,z,pitch,zInner)];
        	f13= fA[f_mem(13,x,y+1,z,pitch,zInner)];
        	f14= fA[f_mem(14,x,y+1,z,pitch,zInner)];
        	f15= fA[f_mem(15,x,y+1,z,pitch,zInner)];
        	f16= fA[f_mem(16,x,y+1,z,pitch,zInner)];
        	f17= fA[f_mem(17,x,y+1,z,pitch,zInner)];
        	f18= fA[f_mem(18,x,y+1,z,pitch,zInner)];

			South_Extrap(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,UMAX);
        }

		boundaries(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z,im);

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		fB[f_mem(0 ,x,y,z,pitch,zInner)] = f0 ;
		fB[f_mem(1 ,x,y,z,pitch,zInner)] = f1 ;
		fB[f_mem(2 ,x,y,z,pitch,zInner)] = f2 ;
		fB[f_mem(3 ,x,y,z,pitch,zInner)] = f3 ;
		fB[f_mem(4 ,x,y,z,pitch,zInner)] = f4 ;
		fB[f_mem(5 ,x,y,z,pitch,zInner)] = f5 ;
		fB[f_mem(6 ,x,y,z,pitch,zInner)] = f6 ;
		fB[f_mem(7 ,x,y,z,pitch,zInner)] = f7 ;
		fB[f_mem(8 ,x,y,z,pitch,zInner)] = f8 ;
		fB[f_mem(9 ,x,y,z,pitch,zInner)] = f9 ;
		fB[f_mem(10,x,y,z,pitch,zInner)] = f10;
		fB[f_mem(11,x,y,z,pitch,zInner)] = f11;
		fB[f_mem(12,x,y,z,pitch,zInner)] = f12;
		fB[f_mem(13,x,y,z,pitch,zInner)] = f13;
		fB[f_mem(14,x,y,z,pitch,zInner)] = f14;
		fB[f_mem(15,x,y,z,pitch,zInner)] = f15;
		fB[f_mem(16,x,y,z,pitch,zInner)] = f16;
		fB[f_mem(17,x,y,z,pitch,zInner)] = f17;
		fB[f_mem(18,x,y,z,pitch,zInner)] = f18;
	}
//	}
}

__global__ void update_bottom(float* gA, float* gB, float* f, float* temp,
							float omega, size_t pitch, int GPU, int zInner)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
    int z = (zInner+2);
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y,GPU*z);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;

	f0 = gA  [j];
	f1 = gA  [buff_mem(1 ,x-1,y  ,pitch)];
	f3 = gA  [buff_mem(3 ,x+1,y  ,pitch)];
	f2 = gA  [buff_mem(2 ,x  ,y-1,pitch)];
	f5 = gA  [buff_mem(5 ,x-1,y-1,pitch)];
	f6 = gA  [buff_mem(6 ,x+1,y-1,pitch)];
	f4 = gA  [buff_mem(4 ,x  ,y+1,pitch)];
	f7 = gA  [buff_mem(7 ,x+1,y+1,pitch)];
	f8 = gA  [buff_mem(8 ,x-1,y+1,pitch)];
	f9 = temp[buff_mem(9 ,x  ,y  ,pitch)];
	f10= temp[buff_mem(10,x-1,y  ,pitch)];
	f11= temp[buff_mem(11,x  ,y-1,pitch)];
	f12= temp[buff_mem(12,x+1,y  ,pitch)];
	f13= temp[buff_mem(13,x  ,y+1,pitch)];
	f14= f   [f_mem   (14,x  ,y  ,0,pitch, zInner)];
	f15= f   [f_mem   (15,x-1,y  ,0,pitch, zInner)];
	f16= f   [f_mem   (16,x  ,y-1,0,pitch, zInner)];
	f17= f   [f_mem   (17,x+1,y  ,0,pitch, zInner)];
	f18= f   [f_mem   (18,x  ,y+1,0,pitch, zInner)];

	if(im == 1 || im ==10){//BB
		gB[buff_mem(0 ,x,y,pitch)] = f0 ;
		gB[buff_mem(1 ,x,y,pitch)] = f3 ;
		gB[buff_mem(2 ,x,y,pitch)] = f4 ;
		gB[buff_mem(3 ,x,y,pitch)] = f1 ;
		gB[buff_mem(4 ,x,y,pitch)] = f2 ;
		gB[buff_mem(5 ,x,y,pitch)] = f7 ;
		gB[buff_mem(6 ,x,y,pitch)] = f8 ;
		gB[buff_mem(7 ,x,y,pitch)] = f5 ;
		gB[buff_mem(8 ,x,y,pitch)] = f6 ;
		gB[buff_mem(9 ,x,y,pitch)] = f14;
		gB[buff_mem(10,x,y,pitch)] = f17;
		gB[buff_mem(11,x,y,pitch)] = f18;
		gB[buff_mem(12,x,y,pitch)] = f15;
		gB[buff_mem(13,x,y,pitch)] = f16;
		gB[buff_mem(14,x,y,pitch)] = f9 ;
		gB[buff_mem(15,x,y,pitch)] = f12;
		gB[buff_mem(16,x,y,pitch)] = f13;
		gB[buff_mem(17,x,y,pitch)] = f10;
		gB[buff_mem(18,x,y,pitch)] = f11;
	}
	else{
        if(im == 100)//north outlet
        {
           	f0 = gA[buff_mem(0 ,x,y-1,pitch)];
        	f1 = gA[buff_mem(1 ,x,y-1,pitch)];
        	f3 = gA[buff_mem(3 ,x,y-1,pitch)];
        	f2 = gA[buff_mem(2 ,x,y-1,pitch)];
        	f5 = gA[buff_mem(5 ,x,y-1,pitch)];
        	f6 = gA[buff_mem(6 ,x,y-1,pitch)];
        	f4 = gA[buff_mem(4 ,x,y-1,pitch)];
        	f7 = gA[buff_mem(7 ,x,y-1,pitch)];
        	f8 = gA[buff_mem(8 ,x,y-1,pitch)];
        	f9 = gA[buff_mem(9 ,x,y-1,pitch)];
        	f10= gA[buff_mem(10,x,y-1,pitch)];
        	f11= gA[buff_mem(11,x,y-1,pitch)];
        	f12= gA[buff_mem(12,x,y-1,pitch)];
        	f13= gA[buff_mem(13,x,y-1,pitch)];
        	f14= gA[buff_mem(14,x,y-1,pitch)];
        	f15= gA[buff_mem(15,x,y-1,pitch)];
        	f16= gA[buff_mem(16,x,y-1,pitch)];
        	f17= gA[buff_mem(17,x,y-1,pitch)];
        	f18= gA[buff_mem(18,x,y-1,pitch)];

			North_Extrap(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,1.0f);
        }
        if(im == 200)//south inlet
        {
           	f0 = gA[buff_mem(0 ,x,y+1,pitch)];
        	f1 = gA[buff_mem(1 ,x,y+1,pitch)];
        	f3 = gA[buff_mem(3 ,x,y+1,pitch)];
        	f2 = gA[buff_mem(2 ,x,y+1,pitch)];
        	f5 = gA[buff_mem(5 ,x,y+1,pitch)];
        	f6 = gA[buff_mem(6 ,x,y+1,pitch)];
        	f4 = gA[buff_mem(4 ,x,y+1,pitch)];
        	f7 = gA[buff_mem(7 ,x,y+1,pitch)];
        	f8 = gA[buff_mem(8 ,x,y+1,pitch)];
        	f9 = gA[buff_mem(9 ,x,y+1,pitch)];
        	f10= gA[buff_mem(10,x,y+1,pitch)];
        	f11= gA[buff_mem(11,x,y+1,pitch)];
        	f12= gA[buff_mem(12,x,y+1,pitch)];
        	f13= gA[buff_mem(13,x,y+1,pitch)];
        	f14= gA[buff_mem(14,x,y+1,pitch)];
        	f15= gA[buff_mem(15,x,y+1,pitch)];
        	f16= gA[buff_mem(16,x,y+1,pitch)];
        	f17= gA[buff_mem(17,x,y+1,pitch)];
        	f18= gA[buff_mem(18,x,y+1,pitch)];

			South_Extrap(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,UMAX);
        }



		boundaries(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z,im);

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		gB[buff_mem(0 ,x,y,pitch)] = f0 ;
		gB[buff_mem(1 ,x,y,pitch)] = f1 ;
		gB[buff_mem(2 ,x,y,pitch)] = f2 ;
		gB[buff_mem(3 ,x,y,pitch)] = f3 ;
		gB[buff_mem(4 ,x,y,pitch)] = f4 ;
		gB[buff_mem(5 ,x,y,pitch)] = f5 ;
		gB[buff_mem(6 ,x,y,pitch)] = f6 ;
		gB[buff_mem(7 ,x,y,pitch)] = f7 ;
		gB[buff_mem(8 ,x,y,pitch)] = f8 ;
		gB[buff_mem(9 ,x,y,pitch)] = f9 ;
		gB[buff_mem(10,x,y,pitch)] = f10;
		gB[buff_mem(11,x,y,pitch)] = f11;
		gB[buff_mem(12,x,y,pitch)] = f12;
		gB[buff_mem(13,x,y,pitch)] = f13;
		gB[buff_mem(14,x,y,pitch)] = f14;
		gB[buff_mem(15,x,y,pitch)] = f15;
		gB[buff_mem(16,x,y,pitch)] = f16;
		gB[buff_mem(17,x,y,pitch)] = f17;
		gB[buff_mem(18,x,y,pitch)] = f18;
	}
//	}
}

__global__ void update_top(float* hA, float* hB, float* f, float* temp,
							float omega, size_t pitch, int GPU, int zInner)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
    int z = (GPU+1)*(zInner+2)-1;//physical coord
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	int im = ImageFcn(x,y,(GPU+1)*(zInner+2)-1);
	float f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18;

	f0 = hA[j];
	f1 = hA  [buff_mem(1 ,x-1,y  ,pitch)];
	f3 = hA  [buff_mem(3 ,x+1,y  ,pitch)];
	f2 = hA  [buff_mem(2 ,x  ,y-1,pitch)];
	f5 = hA  [buff_mem(5 ,x-1,y-1,pitch)];
	f6 = hA  [buff_mem(6 ,x+1,y-1,pitch)];
	f4 = hA  [buff_mem(4 ,x  ,y+1,pitch)];
	f7 = hA  [buff_mem(7 ,x+1,y+1,pitch)];
	f8 = hA  [buff_mem(8 ,x-1,y+1,pitch)];
	f9 = f   [f_mem   (9 ,x  ,y  ,zInner-1,pitch, zInner)];
	f10= f   [f_mem   (10,x-1,y  ,zInner-1,pitch, zInner)];
	f11= f   [f_mem   (11,x  ,y-1,zInner-1,pitch, zInner)];
	f12= f   [f_mem   (12,x+1,y  ,zInner-1,pitch, zInner)];
	f13= f   [f_mem   (13,x  ,y+1,zInner-1,pitch, zInner)];
	f14= temp[buff_mem(14,x  ,y  ,pitch)];
	f15= temp[buff_mem(15,x-1,y  ,pitch)];
	f16= temp[buff_mem(16,x  ,y-1,pitch)];
	f17= temp[buff_mem(17,x+1,y  ,pitch)];
	f18= temp[buff_mem(18,x  ,y+1,pitch)];

	if(im == 1 || im ==10){//BB
		hB[buff_mem(0 ,x,y,pitch)] = f0 ;
		hB[buff_mem(1 ,x,y,pitch)] = f3 ;
		hB[buff_mem(2 ,x,y,pitch)] = f4 ;
		hB[buff_mem(3 ,x,y,pitch)] = f1 ;
		hB[buff_mem(4 ,x,y,pitch)] = f2 ;
		hB[buff_mem(5 ,x,y,pitch)] = f7 ;
		hB[buff_mem(6 ,x,y,pitch)] = f8 ;
		hB[buff_mem(7 ,x,y,pitch)] = f5 ;
		hB[buff_mem(8 ,x,y,pitch)] = f6 ;
		hB[buff_mem(9 ,x,y,pitch)] = f14;
		hB[buff_mem(10,x,y,pitch)] = f17;
		hB[buff_mem(11,x,y,pitch)] = f18;
		hB[buff_mem(12,x,y,pitch)] = f15;
		hB[buff_mem(13,x,y,pitch)] = f16;
		hB[buff_mem(14,x,y,pitch)] = f9 ;
		hB[buff_mem(15,x,y,pitch)] = f12;
		hB[buff_mem(16,x,y,pitch)] = f13;
		hB[buff_mem(17,x,y,pitch)] = f10;
		hB[buff_mem(18,x,y,pitch)] = f11;
	}
	else{
        if(im == 100)//north outlet
        {
           	f0 = hA[buff_mem(0 ,x,y-1,pitch)];
        	f1 = hA[buff_mem(1 ,x,y-1,pitch)];
        	f3 = hA[buff_mem(3 ,x,y-1,pitch)];
        	f2 = hA[buff_mem(2 ,x,y-1,pitch)];
        	f5 = hA[buff_mem(5 ,x,y-1,pitch)];
        	f6 = hA[buff_mem(6 ,x,y-1,pitch)];
        	f4 = hA[buff_mem(4 ,x,y-1,pitch)];
        	f7 = hA[buff_mem(7 ,x,y-1,pitch)];
        	f8 = hA[buff_mem(8 ,x,y-1,pitch)];
        	f9 = hA[buff_mem(9 ,x,y-1,pitch)];
        	f10= hA[buff_mem(10,x,y-1,pitch)];
        	f11= hA[buff_mem(11,x,y-1,pitch)];
        	f12= hA[buff_mem(12,x,y-1,pitch)];
        	f13= hA[buff_mem(13,x,y-1,pitch)];
        	f14= hA[buff_mem(14,x,y-1,pitch)];
        	f15= hA[buff_mem(15,x,y-1,pitch)];
        	f16= hA[buff_mem(16,x,y-1,pitch)];
        	f17= hA[buff_mem(17,x,y-1,pitch)];
        	f18= hA[buff_mem(18,x,y-1,pitch)];

			North_Extrap(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,1.0f);
        }
        if(im == 200)//south inlet
        {
           	f0 = hA[buff_mem(0 ,x,y+1,pitch)];
        	f1 = hA[buff_mem(1 ,x,y+1,pitch)];
        	f3 = hA[buff_mem(3 ,x,y+1,pitch)];
        	f2 = hA[buff_mem(2 ,x,y+1,pitch)];
        	f5 = hA[buff_mem(5 ,x,y+1,pitch)];
        	f6 = hA[buff_mem(6 ,x,y+1,pitch)];
        	f4 = hA[buff_mem(4 ,x,y+1,pitch)];
        	f7 = hA[buff_mem(7 ,x,y+1,pitch)];
        	f8 = hA[buff_mem(8 ,x,y+1,pitch)];
        	f9 = hA[buff_mem(9 ,x,y+1,pitch)];
        	f10= hA[buff_mem(10,x,y+1,pitch)];
        	f11= hA[buff_mem(11,x,y+1,pitch)];
        	f12= hA[buff_mem(12,x,y+1,pitch)];
        	f13= hA[buff_mem(13,x,y+1,pitch)];
        	f14= hA[buff_mem(14,x,y+1,pitch)];
        	f15= hA[buff_mem(15,x,y+1,pitch)];
        	f16= hA[buff_mem(16,x,y+1,pitch)];
        	f17= hA[buff_mem(17,x,y+1,pitch)];
        	f18= hA[buff_mem(18,x,y+1,pitch)];

			South_Extrap(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,UMAX);
        }



		boundaries(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,y,z,im);

		if(MODEL == "MRT")
		mrt_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);
		else if(MODEL == "BGK")
		bgk_collide(f0,f1,f2,f3,f4,f5,f6,f7,f8,f9,f10,f11,f12,f13,f14,f15,f16,f17,f18,omega);

		hB[buff_mem(0 ,x,y,pitch)] = f0 ;
		hB[buff_mem(1 ,x,y,pitch)] = f1 ;
		hB[buff_mem(2 ,x,y,pitch)] = f2 ;
		hB[buff_mem(3 ,x,y,pitch)] = f3 ;
		hB[buff_mem(4 ,x,y,pitch)] = f4 ;
		hB[buff_mem(5 ,x,y,pitch)] = f5 ;
		hB[buff_mem(6 ,x,y,pitch)] = f6 ;
		hB[buff_mem(7 ,x,y,pitch)] = f7 ;
		hB[buff_mem(8 ,x,y,pitch)] = f8 ;
		hB[buff_mem(9 ,x,y,pitch)] = f9 ;
		hB[buff_mem(10,x,y,pitch)] = f10;
		hB[buff_mem(11,x,y,pitch)] = f11;
		hB[buff_mem(12,x,y,pitch)] = f12;
		hB[buff_mem(13,x,y,pitch)] = f13;
		hB[buff_mem(14,x,y,pitch)] = f14;
		hB[buff_mem(15,x,y,pitch)] = f15;
		hB[buff_mem(16,x,y,pitch)] = f16;
		hB[buff_mem(17,x,y,pitch)] = f17;
		hB[buff_mem(18,x,y,pitch)] = f18;
	}
//	}
}

__device__ __inline__ float ld_gb1_cg(const float *addr)
{
    float return_value;
    asm("ld.global.cg.f32 %0, [%1];" : "=f"(return_value) : "l"(addr));
    return return_value;
}

__global__ void initialize_single(float *f, size_t pitch, int GPU_N)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
	int j = x+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
	
	int im = ImageFcn(x,y,z);
	float u,v,w,rho,usqr;
	rho = 1.f;
	u = 0.05f;
	v = UMAX;
	w = 0.0f;

//    if(im == 10 || im == 1){
//    u = 0.0f;
//    v = 0.0f;
//    w = 0.0f;
//    }
	//if(x == 3 ) u = 0.1f;
	usqr = u*u+v*v+w*w;

    if(MODEL == "BGK"){ 
	f[j+0 *pitch*YDIM*ZDIM]= 1.0f/3.0f*(rho-1.5f*usqr);
	f[j+1 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+2 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+3 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	f[j+4 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	f[j+5 *pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	f[j+6 *pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	f[j+7 *pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	f[j+8 *pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	f[j+9 *pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
	f[j+10*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
	f[j+11*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(v+w)-1.5f*usqr);
	f[j+12*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
	f[j+13*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(-v+w)-1.5f*usqr);
	f[j+14*pitch*YDIM*ZDIM]= 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
	f[j+15*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
	f[j+16*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
	f[j+17*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	f[j+18*pitch*YDIM*ZDIM]= 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);   
    }
    else{
                                                                                                                
float f0 = 0.1904761791f*rho+-0.597127747f*usqr                                                 ;
float f1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u                                                   ;
float f2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v                       ;
float f3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u                                                   ;
float f4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v                       ;
float f5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)                            ;
float f6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)                            ;
float f7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)                            ;
float f8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)                            ;
float f9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;
float f10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)                                 ;
float f11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w);
float f12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)                            ;
float f13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w);
float f14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w;
float f15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w)                            ;
float f16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w);
float f17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w)                            ;
float f18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w);

f1 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
f2 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
f3 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
f4 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
f5 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
f6 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
f7 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
f8 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
f9 += -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
f10+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
f11+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
f12+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
f13+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
f14+= -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
f15+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
f16+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
f17+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
f18+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;

f[j+0 *pitch*YDIM*(ZDIM/GPU_N-2)]=f0 ;
f[j+1 *pitch*YDIM*(ZDIM/GPU_N-2)]=f1 ;
f[j+2 *pitch*YDIM*(ZDIM/GPU_N-2)]=f2 ;
f[j+3 *pitch*YDIM*(ZDIM/GPU_N-2)]=f3 ;
f[j+4 *pitch*YDIM*(ZDIM/GPU_N-2)]=f4 ;
f[j+5 *pitch*YDIM*(ZDIM/GPU_N-2)]=f5 ;
f[j+6 *pitch*YDIM*(ZDIM/GPU_N-2)]=f6 ;
f[j+7 *pitch*YDIM*(ZDIM/GPU_N-2)]=f7 ;
f[j+8 *pitch*YDIM*(ZDIM/GPU_N-2)]=f8 ;
f[j+9 *pitch*YDIM*(ZDIM/GPU_N-2)]=f9 ;
f[j+10*pitch*YDIM*(ZDIM/GPU_N-2)]=f10;
f[j+11*pitch*YDIM*(ZDIM/GPU_N-2)]=f11;
f[j+12*pitch*YDIM*(ZDIM/GPU_N-2)]=f12;
f[j+13*pitch*YDIM*(ZDIM/GPU_N-2)]=f13;
f[j+14*pitch*YDIM*(ZDIM/GPU_N-2)]=f14;
f[j+15*pitch*YDIM*(ZDIM/GPU_N-2)]=f15;
f[j+16*pitch*YDIM*(ZDIM/GPU_N-2)]=f16;
f[j+17*pitch*YDIM*(ZDIM/GPU_N-2)]=f17;
f[j+18*pitch*YDIM*(ZDIM/GPU_N-2)]=f18;

    }


	if(x == XDIM-1){
	for(int i = XDIM; i<pitch; i++){
		j = i+y*pitch+z*YDIM*pitch;//index on padded mem (pitch in elements)
		f[j+0 *pitch*YDIM*ZDIM]=0.f;
		f[j+1 *pitch*YDIM*ZDIM]=0.f;
		f[j+2 *pitch*YDIM*ZDIM]=0.f;
		f[j+3 *pitch*YDIM*ZDIM]=0.f;
		f[j+4 *pitch*YDIM*ZDIM]=0.f;
		f[j+5 *pitch*YDIM*ZDIM]=0.f;
		f[j+6 *pitch*YDIM*ZDIM]=0.f;
		f[j+7 *pitch*YDIM*ZDIM]=0.f;
		f[j+8 *pitch*YDIM*ZDIM]=0.f;
		f[j+9 *pitch*YDIM*ZDIM]=0.f;
		f[j+10*pitch*YDIM*ZDIM]=0.f;
		f[j+11*pitch*YDIM*ZDIM]=0.f;
		f[j+12*pitch*YDIM*ZDIM]=0.f;
		f[j+13*pitch*YDIM*ZDIM]=0.f;
		f[j+14*pitch*YDIM*ZDIM]=0.f;
		f[j+15*pitch*YDIM*ZDIM]=0.f;
		f[j+16*pitch*YDIM*ZDIM]=0.f;
		f[j+17*pitch*YDIM*ZDIM]=0.f;
		f[j+18*pitch*YDIM*ZDIM]=0.f;
	}
	}
}

__global__ void initialize_buffer(float *g, size_t pitch)//pitch in elements
{
	int x = threadIdx.x+blockIdx.x*blockDim.x;//coord in linear mem
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int j = x+y*pitch;//index on padded mem (pitch in elements)
	
	float u,v,w,rho,usqr;
	rho = 1.f;
	u = 0.05f;
	v = UMAX;
	w = 0.0f;

	usqr = u*u+v*v+w*w;

    if(MODEL == "BGK"){ 
	g[j+0 *pitch*YDIM]= 1.0f/3.0f*(rho-1.5f*usqr);
	g[j+1 *pitch*YDIM]= 1.0f/18.0f*(rho+3.0f*u+4.5f*u*u-1.5f*usqr);
	g[j+2 *pitch*YDIM]= 1.0f/18.0f*(rho+3.0f*v+4.5f*v*v-1.5f*usqr);
	g[j+3 *pitch*YDIM]= 1.0f/18.0f*(rho-3.0f*u+4.5f*u*u-1.5f*usqr);
	g[j+4 *pitch*YDIM]= 1.0f/18.0f*(rho-3.0f*v+4.5f*v*v-1.5f*usqr);
	g[j+5 *pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(u+v)+4.5f*(u+v)*(u+v)-1.5f*usqr);
	g[j+6 *pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(-u+v)+4.5f*(-u+v)*(-u+v)-1.5f*usqr);
	g[j+7 *pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(-u-v)+4.5f*(-u-v)*(-u-v)-1.5f*usqr);
	g[j+8 *pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(u-v)+4.5f*(u-v)*(u-v)-1.5f*usqr);
	g[j+9 *pitch*YDIM]= 1.0f/18.0f*(rho+3.0f*w+4.5f*w*w-1.5f*usqr);
	g[j+10*pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(u+w)+4.5f*(u+w)*(u+w)-1.5f*usqr);
	g[j+11*pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(v+w)+4.5f*(v+w)*(v+w)-1.5f*usqr);
	g[j+12*pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(-u+w)+4.5f*(-u+w)*(-u+w)-1.5f*usqr);
	g[j+13*pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(-v+w)+4.5f*(-v+w)*(-v+w)-1.5f*usqr);
	g[j+14*pitch*YDIM]= 1.0f/18.0f*(rho-3.0f*w+4.5f*w*w-1.5f*usqr);
	g[j+15*pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(u-w)+4.5f*(u-w)*(u-w)-1.5f*usqr);
	g[j+16*pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(v-w)+4.5f*(v-w)*(v-w)-1.5f*usqr);
	g[j+17*pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(-u-w)+4.5f*(-u-w)*(-u-w)-1.5f*usqr);
	g[j+18*pitch*YDIM]= 1.0f/36.0f*(rho+3.0f*(-v-w)+4.5f*(-v-w)*(-v-w)-1.5f*usqr);   
    }
    else{
                                                                                                                
float f0 = 0.1904761791f*rho+-0.597127747f*usqr                                                 ;
float f1 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*u                                                   ;
float f2 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*v                       ;
float f3 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*u                                                   ;
float f4 = 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*v                       ;
float f5 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+v)                            ;
float f6 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-v)                            ;
float f7 = 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u+v)                            ;
float f8 = 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u-v)                            ;
float f9 = 0.1031746045f*rho+ 0.032375918f*usqr+  0.1666666667f*w;
float f10= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( u+w)                                 ;
float f11= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*( v+w);
float f12= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( u-w)                            ;
float f13= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*( v-w);
float f14= 0.1031746045f*rho+ 0.032375918f*usqr+ -0.1666666667f*w;
float f15= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(u-w)                            ;
float f16= 0.0158730149f*rho+ 0.033572690f*usqr+  0.083333333f*(v-w);
float f17= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(u+w)                            ;
float f18= 0.0158730149f*rho+ 0.033572690f*usqr+ -0.083333333f*(v+w);

f1 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
f2 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
f3 +=  0.055555556f*(2.f*u*u-(v*v+w*w));
f4 += -0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w);
f5 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
f6 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
f7 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+  0.25f*u*v                              ;
f8 +=  0.027777778f*(2.f*u*u-(v*v+w*w))+  0.083333333f*(v*v-w*w)+ -0.25f*u*v                              ;
f9 += -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
f10+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
f11+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w             ;
f12+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
f13+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                        -0.25f*v*w             ;
f14+= -0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                                            ;
f15+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              + -0.25f*u*w;
f16+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      + -0.25f*v*w             ;
f17+=  0.027777778f*(2.f*u*u-(v*v+w*w))+ -0.083333333f*(v*v-w*w)                              +  0.25f*u*w;
f18+= -0.055555556f*(2.f*u*u-(v*v+w*w))                                      +  0.25f*v*w                 ;

g[j+0 *pitch*YDIM]=f0 ;
g[j+1 *pitch*YDIM]=f1 ;
g[j+2 *pitch*YDIM]=f2 ;
g[j+3 *pitch*YDIM]=f3 ;
g[j+4 *pitch*YDIM]=f4 ;
g[j+5 *pitch*YDIM]=f5 ;
g[j+6 *pitch*YDIM]=f6 ;
g[j+7 *pitch*YDIM]=f7 ;
g[j+8 *pitch*YDIM]=f8 ;
g[j+9 *pitch*YDIM]=f9 ;
g[j+10*pitch*YDIM]=f10;
g[j+11*pitch*YDIM]=f11;
g[j+12*pitch*YDIM]=f12;
g[j+13*pitch*YDIM]=f13;
g[j+14*pitch*YDIM]=f14;
g[j+15*pitch*YDIM]=f15;
g[j+16*pitch*YDIM]=f16;
g[j+17*pitch*YDIM]=f17;
g[j+18*pitch*YDIM]=f18;

    }
}

//zMin = minimum zcoord, zNum = number of nodes in z
void WriteResults(float *f, ofstream &output, float omega, int zMin, int zNum)
{
	for(int k = 0; k<zNum; k++){
	for(int i = 0; i<YDIM; i++){
	for(int j = 0; j<XDIM; j++){
 			int index = i*XDIM+j;
            float f0 = f[index+XDIM*YDIM*zNum*0 ];
            float f1 = f[index+XDIM*YDIM*zNum*1 ];
            float f2 = f[index+XDIM*YDIM*zNum*2 ];
            float f3 = f[index+XDIM*YDIM*zNum*3 ];
            float f4 = f[index+XDIM*YDIM*zNum*4 ];
            float f5 = f[index+XDIM*YDIM*zNum*5 ];
            float f6 = f[index+XDIM*YDIM*zNum*6 ];
            float f7 = f[index+XDIM*YDIM*zNum*7 ];
            float f8 = f[index+XDIM*YDIM*zNum*8 ];
            float f9 = f[index+XDIM*YDIM*zNum*9 ];
            float f10= f[index+XDIM*YDIM*zNum*10];
            float f11= f[index+XDIM*YDIM*zNum*11];
            float f12= f[index+XDIM*YDIM*zNum*12];
            float f13= f[index+XDIM*YDIM*zNum*13];
            float f14= f[index+XDIM*YDIM*zNum*14];
            float f15= f[index+XDIM*YDIM*zNum*15];
            float f16= f[index+XDIM*YDIM*zNum*16];
            float f17= f[index+XDIM*YDIM*zNum*17];
            float f18= f[index+XDIM*YDIM*zNum*18];

        	float rho = f0+f1+f2+f3+f4+f5+f6+f7+f8+f9;
        	rho +=f10+f11+f12+f13+f14+f15+f16+f17+f18;
        	float u = f1-f3+f5-f6-f7+f8+f10-f12+f15-f17;
        	float v = f2-f4+f5+f6-f7-f8+f11-f13+f16-f18;
        	float w = f9+f10+f11+f12+f13-f14-f15-f16-f17-f18;
	        float m1  = 19.f*(-f0+ f5+f6+f7+f8+f10+f11+f12+f13+f15+f16+f17+f18   -(u*u+v*v+w*w));
	        float m9  = 2.f*f1+  -  f2+  2.f*f3+  -  f4+ f5+ f6+ f7+ f8+-    f9+ f10+ -2.f*f11+ f12+-2.f*f13+-    f14+ f15+ -2.f*f16+ f17+-2.f*f18  -(2.f*u*u-(v*v+w*w));
	        float m11 =             f2         +     f4+ f5+ f6+ f7+ f8+-    f9+-f10          +-f12         +-    f14+-f15          +-f17         -(v*v-w*w);
	        float m13 =                                  f5+-f6+ f7+-f8                                                                             -u*v;
	        float m14 =                                                                    f11     +-    f13              + -    f16     +     f18  -v*w;
	        float m15 =                                                          f10        + - f12                  +-f15          + f17           -u*w;  
    		float PI11 = -0.026315789f*m1-0.5f *omega*m9;
    		float PI22 = -0.026315789f*m1+0.25f*omega*(m9-3.0f*m11);
    		float PI33 = -0.026315789f*m1+0.25f*omega*(m9+3.0f*m11);
    
    		float PI12 = -1.5f*omega*m13;
    		float PI23 = -1.5f*omega*m14;
    		float PI13 = -1.5f*omega*m15;
    		//float nu0 = ((1.0f/omega)-0.5f)/3.0f;
    		float Smag = sqrt(2.f*(PI11*PI11+PI22*PI22+PI33*PI33+2.f*PI12*PI12+2.f*PI23*PI23+2.f*PI13*PI13));           
			output<<j<<", "<<i<<", "<<zMin+k<<", "<<u<<","<<v<<","<<w<<","<<rho<<","
                  //<<uAv_h[i]<<","<<vAv_h[i]<<", "<<ufluc_h[i]<<","<<vfluc_h[i]<<endl;
                  <<f0<<","<<f1<<", "<<f9<<","<<f18<<endl;
    }}}

}


int main(int argc, char *argv[])
{
	int GPU_N;
	hipGetDeviceCount(&GPU_N);
    GPU_N = 1;
	cout<<"number of GPUs: "<<GPU_N<<endl;


	//int *image_d, *image_h;

	ofstream output;
	ofstream output2;
	string FileName = CASENAME;
	//output.open ("LBM1_out.dat");
	output.open ((FileName+".dat").c_str());
	output2.open ((FileName+".force").c_str());

	//size_t memsize, memsize2;
	size_t pitch = 2;
	while(pitch<XDIM)
		pitch=pitch*2;
	pitch = pitch*sizeof(float);
	size_t pitch_elements = pitch/sizeof(float);

	cout<<"Pitch (in elements): "<<pitch/sizeof(float)<<endl;
	int i, nBlocks;
	float omega, CharLength;

	CharLength = OBSTR1*2.f;

	omega = 1.0f/(3.0f*(UMAX*CharLength/RE)+0.5f);

	cout<<"omega : "<<omega<<endl;
	cout<<"blocksize: "<<BLOCKSIZEX<<"x"<<BLOCKSIZEY<<"x"<<BLOCKSIZEZ<<endl;
	cout<<"grid: "<<XDIM<<"x"<<YDIM<<"x"<<ZDIM<<endl;
	cout<<"TMAX: "<<TMAX<<endl;
	cout<<"Method: "<<METHOD<<endl;
	cout<<"Model: "<<MODEL<<endl;

    int zInner = ZDIM/GPU_N-2; //excluding halo
    //int zGPU = ZDIM/GPU_N;//z nodes per GPU (including halo)

	//nBlocks does not include the halo layers
	nBlocks = ((XDIM+BLOCKSIZEX-1)/BLOCKSIZEX)*((YDIM+BLOCKSIZEY-1)/BLOCKSIZEY)
				*((zInner+BLOCKSIZEZ-1)/BLOCKSIZEZ);
	cout<<"nBlocks:"<<nBlocks<<endl;

    dim3 threads(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	//2 halo layers per GPU (for 2 GPUs)
    dim3 grid(((XDIM+BLOCKSIZEX-1)/BLOCKSIZEX),((YDIM+BLOCKSIZEY-1)/BLOCKSIZEY),(zInner)/BLOCKSIZEZ);
    dim3 g_grid(((XDIM+BLOCKSIZEX-1)/BLOCKSIZEX),((YDIM+BLOCKSIZEY-1)/BLOCKSIZEY),1);
    dim3 h_grid(((XDIM+BLOCKSIZEX-1)/BLOCKSIZEX),((YDIM+BLOCKSIZEY-1)/BLOCKSIZEY),1);


    hipStream_t stream_halo[GPU_N];
    hipStream_t stream_inner[GPU_N];

    //data pointers as 3D array (GPUxCoord)
    float   *f_inner_h[GPU_N],   *g_h[GPU_N],   *h_h[GPU_N];
    float *f_inner_A_d[GPU_N], *g_A_d[GPU_N], *h_A_d[GPU_N];
    float *f_inner_B_d[GPU_N], *g_B_d[GPU_N], *h_B_d[GPU_N];
    float *g_temp[GPU_N], *h_temp[GPU_N];



    //Malloc and Initialize for each GPU
    for(int n = 0; n<GPU_N; n++){
	f_inner_h[n] = (float *)malloc(XDIM*YDIM*zInner*19*sizeof(float));
	g_h      [n] = (float *)malloc(XDIM*YDIM*       19*sizeof(float));
	h_h      [n] = (float *)malloc(XDIM*YDIM*       19*sizeof(float));
	
    hipSetDevice(n);

	hipStreamCreate(&stream_halo[n]);
	hipStreamCreate(&stream_inner[n]);

    for(int m = 0; m<GPU_N; m++){
        if(m != n)
	        hipDeviceEnablePeerAccess(m,0);
    }

	hipMalloc((void **) &f_inner_A_d[n], pitch*YDIM*zInner*19*sizeof(float));
	hipMalloc((void **) &f_inner_B_d[n], pitch*YDIM*zInner*19*sizeof(float));
	hipMalloc((void **) &      g_A_d[n], pitch*YDIM*       19*sizeof(float));
	hipMalloc((void **) &      g_B_d[n], pitch*YDIM*       19*sizeof(float));
	hipMalloc((void **) &      h_A_d[n], pitch*YDIM*       19*sizeof(float));
	hipMalloc((void **) &      h_B_d[n], pitch*YDIM*       19*sizeof(float));
	hipMalloc((void **) &     g_temp[n], pitch*YDIM*       19*sizeof(float));
	hipMalloc((void **) &     h_temp[n], pitch*YDIM*       19*sizeof(float));

	//initialize host f_inner
	for (i = 0; i < XDIM*YDIM*zInner*19; i++)
		f_inner_h[n][i] = 0;
	//initialize host g,h
	for (i = 0; i < XDIM*YDIM*19; i++){
		g_h[n][i] = 0;
		h_h[n][i] = 0;
	}

	hipMemcpy2D(f_inner_A_d[n],pitch,f_inner_h[n],XDIM*sizeof(float),XDIM*sizeof(float),YDIM*zInner*19,hipMemcpyHostToDevice);
	hipMemcpy2D(f_inner_B_d[n],pitch,f_inner_h[n],XDIM*sizeof(float),XDIM*sizeof(float),YDIM*zInner*19,hipMemcpyHostToDevice);
	hipMemcpy2D(      g_A_d[n],pitch,      g_h[n],XDIM*sizeof(float),XDIM*sizeof(float),YDIM*       19,hipMemcpyHostToDevice);
	hipMemcpy2D(      g_B_d[n],pitch,      g_h[n],XDIM*sizeof(float),XDIM*sizeof(float),YDIM*       19,hipMemcpyHostToDevice);
	hipMemcpy2D(      h_A_d[n],pitch,      h_h[n],XDIM*sizeof(float),XDIM*sizeof(float),YDIM*       19,hipMemcpyHostToDevice);
	hipMemcpy2D(      h_B_d[n],pitch,      h_h[n],XDIM*sizeof(float),XDIM*sizeof(float),YDIM*       19,hipMemcpyHostToDevice);

	initialize_single<<<grid  , threads>>>(f_inner_A_d[n],pitch_elements,GPU_N);
	initialize_single<<<grid  , threads>>>(f_inner_B_d[n],pitch_elements,GPU_N);
	initialize_buffer<<<g_grid, threads>>>(      g_A_d[n],pitch_elements);
	initialize_buffer<<<g_grid, threads>>>(      g_B_d[n],pitch_elements);
	initialize_buffer<<<g_grid, threads>>>(      h_A_d[n],pitch_elements);
	initialize_buffer<<<g_grid, threads>>>(      h_B_d[n],pitch_elements);

    }//end Malloc and Initialize

	struct timeval tdr0,tdr1;
	double restime;
	hipDeviceSynchronize();
	gettimeofday (&tdr0,NULL);

    //Time loop
	for(int t = 0; t<TMAX; t+=2){

		//A->B
        for(int n = 0; n<GPU_N; n++){
		hipSetDevice(n);

		hipMemcpyPeerAsync(&h_temp[n][pitch_elements*YDIM*14],n,&g_A_d[   (n+1)%GPU_N][pitch_elements*YDIM*14],   (n+1)%GPU_N,pitch_elements*YDIM*sizeof(float)*5,stream_halo[n]);
		hipMemcpyPeerAsync(&g_temp[n][pitch_elements*YDIM*9 ],n,&h_A_d[abs(n-1)%GPU_N][pitch_elements*YDIM*9 ],abs(n-1)%GPU_N,pitch_elements*YDIM*sizeof(float)*5,stream_halo[n]);

		hipStreamSynchronize(stream_halo[n]);

		update_inner <<<  grid, threads, 0, stream_inner[n]>>>(f_inner_A_d[n],f_inner_B_d[n],      g_A_d[n], h_A_d[n],omega,pitch_elements,n,zInner);
		update_top   <<<h_grid, threads, 0, stream_halo [n]>>>(      h_A_d[n],      h_B_d[n],f_inner_A_d[n],h_temp[n],omega,pitch_elements,n,zInner);
		update_bottom<<<h_grid, threads, 0, stream_halo [n]>>>(      g_A_d[n],      g_B_d[n],f_inner_A_d[n],g_temp[n],omega,pitch_elements,n,zInner);
        }
		hipDeviceSynchronize();

		//B->A
        for(int n = 0; n<GPU_N; n++){
		hipSetDevice(n);

		hipMemcpyPeerAsync(&h_temp[n][pitch_elements*YDIM*14],n,&g_B_d[   (n+1)%GPU_N][pitch_elements*YDIM*14],   (n+1)%GPU_N,pitch_elements*YDIM*sizeof(float)*5,stream_halo[n]);
		hipMemcpyPeerAsync(&g_temp[n][pitch_elements*YDIM*9 ],n,&h_B_d[abs(n-1)%GPU_N][pitch_elements*YDIM*9 ],abs(n-1)%GPU_N,pitch_elements*YDIM*sizeof(float)*5,stream_halo[n]);

		hipStreamSynchronize(stream_halo[n]);

		update_inner <<<  grid, threads, 0, stream_inner[n]>>>(f_inner_B_d[n],f_inner_A_d[n],      g_B_d[n], h_B_d[n],omega,pitch_elements,n,zInner);
		update_top   <<<h_grid, threads, 0, stream_halo [n]>>>(      h_B_d[n],      h_A_d[n],f_inner_B_d[n],h_temp[n],omega,pitch_elements,n,zInner);
		update_bottom<<<h_grid, threads, 0, stream_halo [n]>>>(      g_B_d[n],      g_A_d[n],f_inner_B_d[n],g_temp[n],omega,pitch_elements,n,zInner);
        }
		hipDeviceSynchronize();



    }//end Time loop

	hipDeviceSynchronize();
	gettimeofday (&tdr1,NULL);
	timeval_subtract (&restime, &tdr1, &tdr0);
	int Nodes;
	Nodes = XDIM*YDIM*ZDIM;
	cout<<"Time taken for main kernel: "<<restime<<" ("
			<<double(Nodes*double(TMAX/1000000.f))/restime<<"MLUPS)\n";


	output<<"VARIABLES = \"X\",\"Y\",\"Z\",\"u\",\"v\",\"w\",\"rho\",\"uAv\",\"vAv\",\"ufluc\",\"vfluc\"\n";
	output<<"ZONE F=POINT, I="<<XDIM<<", J="<<YDIM<<", K="<<ZDIM<<"\n";

    //D2H Memcpy and write results
    for(int n = 0; n<GPU_N; n++){
    hipSetDevice(n);

	hipMemcpy2D(f_inner_h[n],XDIM*sizeof(float),f_inner_A_d[n],pitch,XDIM*sizeof(float),YDIM*zInner*19,hipMemcpyDeviceToHost);
	hipMemcpy2D(      g_h[n],XDIM*sizeof(float),      g_A_d[n],pitch,XDIM*sizeof(float),YDIM*       19,hipMemcpyDeviceToHost);
	hipMemcpy2D(      h_h[n],XDIM*sizeof(float),      h_A_d[n],pitch,XDIM*sizeof(float),YDIM*       19,hipMemcpyDeviceToHost);

    //Write results
    WriteResults(      g_h[n],output,omega,ZDIM/GPU_N*n      ,1     );
    WriteResults(f_inner_h[n],output,omega,ZDIM/GPU_N*n+1    ,zInner);
    WriteResults(      h_h[n],output,omega,ZDIM/GPU_N*(n+1)-1,1     );

	hipFree(f_inner_A_d[n]);
	hipFree(f_inner_B_d[n]);
	hipFree(      g_A_d[n]);
	hipFree(      g_B_d[n]);
	hipFree(      h_A_d[n]);
	hipFree(      h_B_d[n]);
	hipFree(     g_temp[n]);
	hipFree(     h_temp[n]);
    }//end write results

	return(0);

}
